#include "hip/hip_runtime.h"
/*
 * c_instance.cu
 *
 *  Created on: Feb 8, 2012
 *      Author: tkalbitz
 */

#include <assert.h>
#include <math.h>
#include <time.h>

#include "c_instance.h"
#include "c_setup.h"

void init_rnd_generator(struct c_instance& inst, int seed)
{
	hiprandState *rnd_states;

	int count = 320;

	CUDA_CALL(hipMalloc(&rnd_states, count * BLOCKS *
			sizeof(hiprandState)));

	inst.rnd_states = rnd_states;

	setup_c_rnd_kernel<<<1, count>>>(inst, seed);
	CUDA_CALL(hipGetLastError());
}

void set_num_matrices(struct c_instance& inst)
{
	int m = INT_MIN;
	for(size_t i = 0; i < inst.rules_len; i++)
		m = max(m, inst.rules[i]);

	inst.num_matrices = m + 1; /* matrices are zero based */
}

void alloc_instance_mem(struct c_instance& inst)
{
	assert(inst.num_matrices != 0);

	const size_t ilen = inst.itotal * sizeof(double);
	const size_t slen = inst.stotal * sizeof(double);
	const size_t tlen = BLOCKS * inst.width_per_inst * sizeof(double);
	const size_t reslen = inst.icount * inst.mdim * inst.mdim * BLOCKS *
				sizeof(double);

	CUDA_CALL(hipMalloc(&(inst.tmp),        tlen));
	CUDA_CALL(hipMalloc(&(inst.tmprat),     BLOCKS * sizeof(double)));
	CUDA_CALL(hipMalloc(&(inst.instances),  ilen));
	CUDA_CALL(hipMalloc(&(inst.sinstances), slen));
	CUDA_CALL(hipMalloc(&(inst.best), BLOCKS * sizeof(*inst.best)));
	CUDA_CALL(hipMalloc(&(inst.best_idx), BLOCKS * sizeof(*inst.best_idx)));

	const size_t ratlen = BLOCKS * inst.icount * sizeof(*inst.rating);
	CUDA_CALL(hipMalloc(&(inst.rating), ratlen));
	CUDA_CALL(hipMalloc(&(inst.srating), BLOCKS * sizeof(*inst.srating)));

	CUDA_CALL(hipMalloc(&(inst.res), reslen));
}

void c_inst_init(struct c_instance& inst, int matrix_width)
{
	inst.mdim = matrix_width;
	set_num_matrices(inst);

	inst.width_per_matrix = inst.mdim * inst.mdim;
	inst.width_per_inst = inst.num_matrices * inst.mdim * inst.mdim;

	inst.itotal = inst.width_per_inst * inst.icount * BLOCKS;
	inst.stotal = inst.width_per_inst * BLOCKS;

	alloc_instance_mem(inst);
	init_rnd_generator(inst, (int)time(0));
}

void c_inst_cleanup(struct c_instance& inst,
		    struct c_instance* dev_inst)
{
	if(dev_inst != NULL)
		hipFree(dev_inst);

	hipFree(inst.rnd_states);
	hipFree(inst.res);
	hipFree(inst.instances);
	hipFree(inst.sinstances);
	hipFree(inst.rating);
	hipFree(inst.srating);
	hipFree(inst.best);
	hipFree(inst.best_idx);
	hipFree(inst.tmp);
	hipFree(inst.tmprat);
}

struct c_instance* c_inst_create_dev_inst(struct c_instance& inst,
					  int** dev_rules)
{
	struct c_instance *dev_inst;
	int *rules = inst.rules;
	int *tmp_dev_rules;
	CUDA_CALL(hipMalloc(&tmp_dev_rules, inst.rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(tmp_dev_rules,  rules, inst.rules_len * sizeof(int),
					hipMemcpyHostToDevice));

	inst.rules = tmp_dev_rules;
	CUDA_CALL(hipMalloc(&dev_inst, sizeof(*dev_inst)));
	CUDA_CALL(hipMemcpy(dev_inst,  &inst, sizeof(*dev_inst),
					hipMemcpyHostToDevice));
	if(dev_rules != NULL)
		*dev_rules = tmp_dev_rules;

	return dev_inst;
}
