#include "hip/hip_runtime.h"
/*
 * c_instance.cu
 *
 *  Created on: Feb 8, 2012
 *      Author: tkalbitz
 */

#include <assert.h>
#include <math.h>
#include <time.h>

#include "c_instance.h"
#include "c_setup.h"

void init_rnd_generator(struct c_instance& inst, int seed)
{
	hiprandState *rnd_states;

	CUDA_CALL(hipMalloc(&rnd_states, inst.scount * BLOCKS *
			sizeof(hiprandState)));

	inst.rnd_states = rnd_states;

	const int threads = min(inst.scount, 512);
	setup_c_rnd_kernel<<<1, threads>>>(inst, seed);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
}

void set_num_matrices(struct c_instance& inst)
{
	int m = INT_MIN;
	for(size_t i = 0; i < inst.rules_len; i++)
		m = max(m, inst.rules[i]);

	inst.num_matrices = m + 1; /* matrices are zero based */
}

void alloc_instance_mem(struct c_instance& inst)
{
	assert(inst.num_matrices != 0);

	const size_t ilen = inst.itotal * sizeof(double);
	const size_t slen = inst.stotal * sizeof(double);
	const size_t tlen = BLOCKS * inst.width_per_inst * sizeof(double);
	const size_t reslen = inst.scount * inst.mdim * inst.mdim * BLOCKS *
			      sizeof(double);

	CUDA_CALL(hipMalloc(&(inst.tmp),        tlen));
	CUDA_CALL(hipMalloc(&(inst.instances),  ilen));
	CUDA_CALL(hipMalloc(&(inst.sinstances), slen));
	CUDA_CALL(hipMalloc(&(inst.best), BLOCKS * sizeof(*inst.best)));

	const size_t ratlen = BLOCKS * inst.icount * sizeof(*inst.rating);
	CUDA_CALL(hipMalloc(&(inst.rating), ratlen));

	const size_t sratlen = BLOCKS * inst.scount * sizeof(*inst.srating);
	CUDA_CALL(hipMalloc(&(inst.srating), sratlen));

	CUDA_CALL(hipMalloc(&(inst.res), reslen));
}

void c_inst_init(struct c_instance& inst, int matrix_width)
{
	inst.mdim = matrix_width;
	set_num_matrices(inst);

	inst.width_per_matrix = inst.mdim * inst.mdim;
	inst.width_per_inst = inst.num_matrices * inst.mdim * inst.mdim;

	inst.itotal = inst.width_per_inst * inst.icount * BLOCKS;
	inst.stotal = inst.width_per_inst * inst.scount * BLOCKS;

	alloc_instance_mem(inst);
	init_rnd_generator(inst, (int)time(0));
}

void c_inst_cleanup(struct c_instance& inst,
		    struct c_instance* dev_inst)
{
	if(dev_inst != NULL)
		hipFree(dev_inst);

	hipFree(inst.rnd_states);
	hipFree(inst.res);
	hipFree(inst.instances);
	hipFree(inst.sinstances);
	hipFree(inst.rating);
	hipFree(inst.srating);
	hipFree(inst.best);
	hipFree(inst.tmp);
}

struct c_instance* c_inst_create_dev_inst(struct c_instance& inst,
					  int** dev_rules)
{
	struct c_instance *dev_inst;
	int *rules = inst.rules;
	int *tmp_dev_rules;
	CUDA_CALL(hipMalloc(&tmp_dev_rules, inst.rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(tmp_dev_rules,  rules, inst.rules_len * sizeof(int),
					hipMemcpyHostToDevice));

	inst.rules = tmp_dev_rules;
	CUDA_CALL(hipMalloc(&dev_inst, sizeof(*dev_inst)));
	CUDA_CALL(hipMemcpy(dev_inst,  &inst, sizeof(*dev_inst),
					hipMemcpyHostToDevice));
	if(dev_rules != NULL)
		*dev_rules = tmp_dev_rules;

	return dev_inst;
}
