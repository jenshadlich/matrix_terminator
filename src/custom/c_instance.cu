#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, 2012 Tobias Kalbitz <tobias.kalbitz@googlemail.com>
 *
 * All rights reserved. This program and the accompanying materials
 * are made available under the terms of the GNU Public License v2.0
 * which accompanies this distribution, and is available at
 * http://www.gnu.org/licenses/old-licenses/gpl-2.0.html
 */

#include <assert.h>
#include <math.h>
#include <time.h>

#include "c_instance.h"
#include "c_setup.h"

void init_rnd_generator(struct c_instance& inst, int seed)
{
	hiprandState *rnd_states;

	int count = 320;

	CUDA_CALL(hipMalloc(&rnd_states, count * BLOCKS *
			sizeof(hiprandState)));

	inst.rnd_states = rnd_states;

	setup_c_rnd_kernel<<<1, count>>>(inst, seed);
	CUDA_CALL(hipGetLastError());
}

void set_num_matrices(struct c_instance& inst)
{
	int m = INT_MIN;
	for(size_t i = 0; i < inst.rules_len; i++)
		m = max(m, inst.rules[i]);

	inst.num_matrices = m + 1; /* matrices are zero based */
}

void alloc_instance_mem(struct c_instance& inst)
{
	assert(inst.num_matrices != 0);

	const size_t ilen = inst.itotal * sizeof(float);

	CUDA_CALL(hipMalloc(&(inst.instances),  ilen));
	CUDA_CALL(hipMalloc(&(inst.best), BLOCKS * sizeof(*inst.best)));
	CUDA_CALL(hipMalloc(&(inst.best_idx), BLOCKS * sizeof(*inst.best_idx)));

	const size_t ratlen = BLOCKS * inst.icount * sizeof(*inst.rating);
	CUDA_CALL(hipMalloc(&(inst.rating), ratlen));
}

void c_inst_init(struct c_instance& inst, int matrix_width)
{
	inst.mdim = matrix_width;
	set_num_matrices(inst);

	inst.width_per_matrix = inst.mdim * inst.mdim;
	inst.width_per_inst = inst.num_matrices * inst.mdim * inst.mdim;

	inst.itotal = inst.width_per_inst * inst.icount * BLOCKS;

	alloc_instance_mem(inst);
	init_rnd_generator(inst, (int)time(0));
}

void c_inst_cleanup(struct c_instance& inst)
{
	hipFree(inst.rnd_states);
	hipFree(inst.instances);
	hipFree(inst.rating);
	hipFree(inst.best);
	hipFree(inst.best_idx);
}

int* c_create_dev_rules(struct c_instance& inst)
{
	int *rules = inst.rules;
	int *tmp_dev_rules;
	CUDA_CALL(hipMalloc(&tmp_dev_rules, inst.rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(tmp_dev_rules,  rules, inst.rules_len * sizeof(int),
			     hipMemcpyHostToDevice));

	return tmp_dev_rules;
}
