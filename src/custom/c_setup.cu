#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, 2012 Tobias Kalbitz <tobias.kalbitz@googlemail.com>
 *
 * All rights reserved. This program and the accompanying materials
 * are made available under the terms of the GNU Public License v2.0
 * which accompanies this distribution, and is available at
 * http://www.gnu.org/licenses/old-licenses/gpl-2.0.html
 */

#include <float.h>

#include "c_config.h"
#include "c_setup.h"

/* calculate the thread id for the current block topology */
__device__ inline int get_thread_id() {
	const int uniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
	const int uniqueThreadIndex =
			uniqueBlockIndex * blockDim.y * blockDim.x +
			threadIdx.y * blockDim.x + threadIdx.x;
	return uniqueThreadIndex;
}

/* calculate the thread id for the current block topology */
__device__ inline int get_max_thread_id() {
	const int uniqueBlockIndex =
			(gridDim.y - 1) * gridDim.x + (gridDim.x - 1);
	const int uniqueThreadIndex =
			uniqueBlockIndex * blockDim.y * blockDim.x +
			(blockDim.y - 1) * blockDim.x + (blockDim.x - 1);
	return uniqueThreadIndex;
}

__device__ static float new_value(struct c_instance& inst,
					   hiprandState* const rnd_state)
{
	/* we want to begin with small numbers */
	const int tmp = (inst.parent_max > 10) ? 10 : (int)inst.parent_max;
	const int rnd_val = (hiprand(rnd_state) % (tmp - 1)) + 1;
	int factor = (int)(rnd_val / inst.delta);
	if((factor * inst.delta) < 1.0)
		factor++;

	const float val = factor * inst.delta;
	if(val < 1.0)
		return 1.0;

	return val;
}

__global__ void setup_c_rnd_kernel(struct c_instance inst,
				   const int seed)
{
	const int end = 320 * BLOCKS;
	for(int i = tx; i < end; i+= blockDim.x)
		hiprand_init(seed + i, i, 0, &(inst.rnd_states[i]));
}

__global__ void patch_matrix_kernel(struct c_instance inst)
{
	float* ind = inst.instances + bx * inst.width_per_inst * inst.icount;
	const int count = inst.num_matrices * inst.icount;

	for(int i = 0; i < count; i++) {
		float* matrix = ind + i * inst.width_per_matrix;
		matrix[tx * inst.mdim] = 0;
		matrix[(inst.mdim - 1) * inst.mdim + tx] = 0;
		matrix[0] = 1;
		matrix[(inst.mdim - 1) * inst.mdim + (inst.mdim - 1)] = 1;
	}

}

__global__ void
setup_instances_kernel(struct c_instance inst)
{
	const int id = get_thread_id();
	const int max_id = get_max_thread_id();
	hiprandState rnd = inst.rnd_states[id];

	const int max1 = (int)inst.parent_max;
	const float delta = inst.delta;
	int x;
	float tmp;


	for(x = id; x < inst.itotal; x += max_id) {
		tmp = hiprand(&rnd) % 2;
		tmp = __fmul_rn(__float2uint_rn(tmp / delta), delta);
		inst.instances[x] = tmp;
	}

	__syncthreads();

	const int matrices = inst.num_matrices * inst.icount * BLOCKS;

	if(inst.cond_left == COND_UPPER_LEFT) {
		for(x = id; x < matrices; x += max_id) {
			const int matrix = x * inst.width_per_matrix;
                        inst.instances[matrix] = new_value(inst, &rnd);
		}
	} else if(inst.cond_left == COND_UPPER_RIGHT) {
		for(x = id; x < matrices; x += max_id) {
			const int matrix = x * inst.width_per_matrix +
					inst.mdim - 1;
                        inst.instances[matrix] = new_value(inst, &rnd);
		}
	} else if(inst.cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
		for(x = id; x < matrices; x += max_id) {
			const int matrix1 = x * inst.width_per_matrix;
			const int matrix2 = (x + 1) * inst.width_per_matrix - 1;
                        inst.instances[matrix1] = new_value(inst, &rnd);
                        inst.instances[matrix2] = new_value(inst, &rnd);
		}
	}
	inst.rnd_states[id] = rnd;
}


__global__ void setup_best_kernel(struct c_instance inst)
{
	inst.best[tx] = FLT_MAX;
}

