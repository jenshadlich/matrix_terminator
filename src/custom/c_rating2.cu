#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "c_rating.h"
#include "c_instance.h"


#define RIDX(cy, cx) ((cy) * mdim + (cx))
#define RES(cy, cx)  res[RIDX(cy, cx)]
#define TRES(cy, cx) slhs[RIDX(cy, cx)]

__shared__ double sind[2 * MATRIX_WIDTH * MATRIX_WIDTH];

__shared__ double slhs[MATRIX_WIDTH * MATRIX_WIDTH];
__shared__ double res[MATRIX_WIDTH * MATRIX_WIDTH];

__shared__ volatile double shrd_rating;
__shared__ double matrix_form;

__shared__ double old_rat;
__shared__ hiprandState rnd;

__shared__ int irules[100];
__shared__ int* rend;

template<int mdim>
__device__ void eval_set_res_matrix_to_identity(const struct c_instance& inst)
{
	if(tx != ty) {
		RES(ty, tx) = 0.;
	} else {
		RES(ty, tx) = 1.;
	}
}

template<int mdim>
__device__ inline void eval_copy_matrix_to_res(const struct c_instance& inst,
		    	    	    	       const int matrix)
{
	const int tid = RIDX(ty, tx);
	const int mat = matrix * mdim * mdim;
	res[tid] = sind[mat + tid];
}

template<int mdim>
__device__ void  eval_mul_inplace(const struct c_instance& inst, const int matrix)
{
	double y, t;
	double c = 0;
	double sum = 0;

	const int mat = matrix * mdim * mdim;

	/* result rows */
	for(int i = 0; i < mdim; i++) {
		y = __dmul_rn(RES(ty, i), sind[mat + RIDX(i, tx)]) - c;
		t = __dadd_rn(sum, y);
		c = (t - sum) - y;
		sum = t;
	}

	__syncthreads();
	RES(ty, tx) = sum;
	__syncthreads();
}

template<int mdim>
__device__ const int* eval_interpret_rule(const struct c_instance& inst,
				    	  const int              * rule)
{
	if(*rule == MUL_SEP)
		return rule;

	/*
	 * all multiplications are inplace,
	 * so we copy the first matrix to our result
	 */
	eval_copy_matrix_to_res<mdim>(inst, *rule);
	rule++;

	__syncthreads();

	for(; *rule != MUL_SEP; rule++) {
		eval_mul_inplace<mdim>(inst, *rule);
	}

	return rule;
}

template<int mdim>
__device__ void c_result_rating(const struct c_instance& inst)
{
	double rating = 0.;

        if(ty == 0 && tx == 0) {
        	const double penalty = 1e6;
        	const int rows = mdim - 1;

                switch(inst.cond_right) {
                case COND_UPPER_LEFT:
                        if((TRES(0, 0) - RES(0, 0)) < 1.f)
                                rating += penalty;
                        break;
                case COND_UPPER_RIGHT:
                        if((TRES(0, rows) - RES(0, rows)) < 1.f)
                                rating += penalty;
                        break;
                case COND_UPPER_LEFT_LOWER_RIGHT:
                        if((TRES(0, 0) - RES(0, 0)) < 1.f)
                                rating += penalty;

                        if((TRES(rows, rows) - RES(rows, rows)) < 1.f)
                                rating += penalty;
                        break;
                default:
                        rating += 2*penalty;
                        break;
                }

                if(inst.match == MATCH_ANY) {
                        if(rating == 0.)
                                matrix_form = 0.;

                        rating = 0.;
                }
        }
	__syncthreads();
	// keep only negative numbers
//	if(min(TRES(ty, tx) - (RES(ty, tx)), 0.) == 0.)
//		RES(ty, tx) = 0;
//	else
//		RES(ty, tx) = (RES(ty, tx) + 1) / (TRES(ty, tx) + 1);

	const double a =  RES(ty, tx);
	const double b = TRES(ty, tx);

	RES(ty, tx) = a > b ? (a * a - b * b) : 0.;
//	RES(ty, tx) = fabs(min(b - a, 0.));
//	RES(ty, tx) = __dmul_rn(RES(ty, tx), RES(ty, tx));

	__syncthreads();

	double c = 0.0;
	double y, t;
	double sum;

	//only lines are processed
	if(tx == 0) {
		sum = 0.;

		for(int i = 0; i < mdim; i++) {
			y = RES(ty, i) - c;
			t = sum + y;
			c = (t - sum) - y;
			sum = t;
		}

		RES(ty, 0) = sum;
	}
	__syncthreads();

	c = 0.0;
	if(tx == 0 && ty == 0) {
		for(int i = 0; i < mdim; i++) {
			y = RES(i, 0) - c;
			t = rating + y;
			c = (t - rating) - y;
			rating = t;
		}

		shrd_rating += rating;
	}
	__syncthreads();
}

template<int mdim>
__device__ void c_calc_res(const struct c_instance& inst)
{
	const int* rules = irules;

	if(tx == 0 && ty == 0) {
		shrd_rating = 0.;
		matrix_form = 1e9;
	}

	__syncthreads();

	do {
		eval_set_res_matrix_to_identity<mdim>(inst);

		rules++;
		rules = eval_interpret_rule<mdim>(inst, rules);

		__syncthreads();
		TRES(ty, tx) = RES(ty, tx);
		__syncthreads();
		eval_set_res_matrix_to_identity<mdim>(inst);
		__syncthreads();

		rules++;
		rules = eval_interpret_rule<mdim>(inst, rules);
		__syncthreads();

		c_result_rating<mdim>(inst);
		__syncthreads();
	} while(rules != rend);

	__syncthreads();

	if(tx == 0 && ty == 0) {
		if(inst.match == MATCH_ANY)
			shrd_rating += matrix_form;
	}
}

template<int mnum, int mdim>
__device__ void copy_to_child(struct c_instance& inst)
{
	__shared__ int child;
	const int bbx = bx;
	double* const rat = inst.rating + bbx * inst.icount;
	const int iwidth = mnum*mdim*mdim;

	if(tx == 0 && ty == 0) {
		child = hiprand(&rnd) % inst.icount;

		if(old_rat < rat[child]) {
			if(old_rat < inst.best[bbx]) {
				inst.best[bbx] = old_rat;
				inst.best_idx[bbx] = child;
			}

			rat[child] = old_rat;
			child = (bbx * inst.icount + child) * iwidth;
		} else {
			child = -1;
		}
	}
	__syncthreads();

	if(child == -1)
		return;

	double* dest = inst.instances + child;
	for(int i = RIDX(ty, tx); i < iwidth; i += mdim*mdim) {
		dest[i] = sind[i];
	}
}

template<int mnum, int mdim>
__device__ void copy_parent(struct c_instance& inst)
{
	const int iwidth = mnum*mdim*mdim;

	__shared__ int parent;
	if(tx == 0 && ty == 0) {
		parent = hiprand(&rnd) % inst.icount;
		parent = (blockIdx.x * inst.icount + parent) * iwidth;
	}
	__syncthreads();
	double* src = inst.instances + parent;

	for(int i = RIDX(ty, tx); i < iwidth; i += mdim*mdim) {
		sind[i] = src[i];
	}
}

template<int mnum, int mdim>
__device__  void path_mutate_p1(struct c_instance& inst,
		                int3* stack, unsigned int* top)
{
	const int* rules = irules;
	const int iwidth = mnum*mdim*mdim;

	int pos;
	int cur_rule = 0;
	int3 entry;

	stack += bx * inst.rules_count * iwidth;
	top += bx;

	if(tx == 0 && ty == 0) {
		atomicExch(top, 0);

		entry.x = 0;
		entry.y = 0;
		entry.z = 0;
		stack[0] = entry;
	}

	__syncthreads();

	const int rows = mdim - 1;
	int special = 0;

	if(inst.cond_right == COND_UPPER_LEFT && ty == 0 && tx == 0)
		special = 1;
	if(inst.cond_right == COND_UPPER_RIGHT && ty == 0 && tx == rows)
		special = 1;
	if(inst.cond_right == COND_UPPER_LEFT_LOWER_RIGHT &&
		((ty == 0 && tx == 0) || (ty == rows && tx == rows)))
		special = 1;

	do {
		eval_set_res_matrix_to_identity<mdim>(inst);

		rules++;
		rules = eval_interpret_rule<mdim>(inst, rules);

		__syncthreads();
		TRES(ty, tx) = RES(ty, tx);
		__syncthreads();
		eval_set_res_matrix_to_identity<mdim>(inst);
		__syncthreads();

		rules++;
		rules = eval_interpret_rule<mdim>(inst, rules);
		__syncthreads();

		entry.x = tx;
		entry.y = ty;
		entry.z = cur_rule;
		const double lhs = TRES(ty, tx);
		const double rhs = RES(ty, tx);

		const int ok = special ? ((lhs - rhs) >= 1.) : lhs >= rhs;
		if(!ok) {
			pos = atomicAdd(top, 1);
			stack[pos] = entry;
		}

		cur_rule++;
		__syncthreads();
	} while(rules != rend);
}

template<int mnum, int mdim>
__device__ void path_mutate_p2(struct c_instance& inst, int3* stack,
		                      unsigned int* top)
{
	const int iwidth = mnum*mdim*mdim;

	const int tid = bx;
	const int* rules = irules;

	int cur_rule = 0;

	stack += tid * inst.rules_count * iwidth;
	top += tid;

	const int chosen = (*top < 2 ? 0 : hiprand(&rnd) % *top);
	int3 entry = stack[chosen];
	int l = entry.y;
	int r = entry.x;
	int goal;

	/* at least go to the first entry */
	rules++;

	/* we have to jump to the rule for that entry */
	while(cur_rule != entry.z) {
		while(*rules != MUL_SEP)
			rules++;

		rules++;

		while(*rules != MUL_SEP)
			rules++;

		rules++;
		cur_rule++;
	}

	/* put new weights on the path */
	for(; *rules != MUL_SEP; rules++) {
		goal = *(rules+1) < 0 ? r : 1 + hiprand(&rnd) % (mdim - 2);
		double* pos = sind + (*rules) * iwidth + l * mdim + goal;
		*pos = max(*pos + inst.delta, 1.);
		l = goal;
	}
}

template<int mnum, int mdim>
__global__ void all_in_one_kernel(struct c_instance inst, int3* stack,
                		  unsigned int* top, const int lucky)
{
	const int bbx = blockIdx.x;

	/* mutation */
	double old_val;
	int    mut_pos;

	if(tx == 0 && ty == 0) {
		rnd = inst.rnd_states[bbx];
		rend = irules + inst.rules_len - 1;

	}

	/* cahing of rules to speed up access */
	for(int i = RIDX(ty, tx); i < inst.rules_len; i += mdim*mdim)
		irules[i] = inst.rules[i];

	copy_parent<mnum, mdim>(inst);
	__syncthreads();

	path_mutate_p1<mnum, mdim>(inst, stack, top);
	__syncthreads();

	if(tx == 0 && ty == 0)
		path_mutate_p2<mnum, mdim>(inst, stack, top);
	__syncthreads();

	c_calc_res<mdim>(inst);
	if(tx == 0 && ty == 0)
		old_rat = shrd_rating;
	__syncthreads();

	for(int steps = 0; steps < lucky; steps++) {

		if(tx == 0 && ty == 0) {
			const int mat = hiprand(&rnd) % mnum;
			const int row = hiprand(&rnd) % (mdim -1);
			const int col = 1 + hiprand(&rnd) % (mdim -1);
			mut_pos = mat*mdim*mdim + row * mdim + col;
			old_val = sind[mut_pos];
			sind[mut_pos] = max(old_val - inst.delta, 0.);
		}
		__syncthreads();

		/* rating of mutated kernel */
		c_calc_res<mdim>(inst);
		__syncthreads();

		/* copy back */
		if(tx == 0 && ty == 0) {
			const int luck = hiprand(&rnd) % lucky;

			if(shrd_rating > old_rat && luck) {
				sind[mut_pos] = old_val;
			} else {
				old_rat = shrd_rating;
			}
		}
	}

	if(old_rat == inst.tmprat[bbx])
		return;

	copy_to_child<mnum, mdim>(inst);
	inst.rnd_states[bbx] = rnd;
}
