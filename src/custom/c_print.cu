#include "c_print.h"
#include "ya_malloc.h"

void print_matrix_pretty(FILE* f, struct c_instance& inst, int block, int bpos)
{
	int width = inst.itotal * sizeof(double);
	double* global_cpy = (double*)ya_malloc(width);
	memset(global_cpy, 1, width);

	CUDA_CALL(hipMemcpy(global_cpy, inst.instances, width,
			hipMemcpyDeviceToHost));

	int block_offset = inst.width_per_inst * inst.icount * block;
	double* ptr = global_cpy + block_offset + bpos * inst.width_per_inst;

//	printf("{");
//	for(int i = 0; i < inst.width_per_inst; i++)
//		printf("%.2e, ", ptr[i]);
//
//	printf("}\n");

	for(int m = 0; m < inst.num_matrices; m++) {
		char matrix = 'A' + m;
		fprintf(f, "%c: matrix(\n", matrix);

		for (int h = 0; h < inst.mdim; h++) {
			int pos = m * inst.width_per_matrix +
				  h * inst.mdim;
			fprintf(f, "[ ");

			for (int w = 0; w < inst.mdim - 1; w++) {
				fprintf(f, "%10.9e, ", ptr[pos + w]);
			}

			fprintf(f, "%10.9e ]", ptr[pos + inst.mdim - 1]);

			if(h < (inst.mdim - 1))
				fprintf(f, ",");
			fprintf(f, "\n");
		}
		fprintf(f, ");\n%c: factor(%c);\n\n", matrix, matrix);
	}

	fprintf(f, "\n");
	free(global_cpy);
}

void print_rules(FILE* f, struct c_instance& inst)
{
	bool mul_sep_count = false;
	bool old_mul_sep_count = true;

	for(uint32_t i = 1; i < inst.rules_len; i++) {

		if(old_mul_sep_count != mul_sep_count) {
			if(mul_sep_count == false)
				fprintf(f, "ratsimp(factor(");
			old_mul_sep_count = mul_sep_count;
		}

		if(inst.rules[i] == MUL_SEP) {
			if(mul_sep_count == false)
				fprintf(f, "ident(%d)-", inst.mdim);
			else
				fprintf(f, "ident(%d)));\n\n", inst.mdim);

			mul_sep_count = !mul_sep_count;
		} else {
				fprintf(f, "%c.", 'A' + inst.rules[i]);
		}
	}

	fprintf(f, "\n");
}
