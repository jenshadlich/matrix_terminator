#include "hip/hip_runtime.h"
// this program searches a matrix impretation
// over the naturals that is compatible with
// z001 = ( RULES a a b b -> b b b a a a) .

// compile: gcc -O6 -std=gnu9x -o matrix matrix.c
// run (example): ./matrix 5 1000 100
// should give a result within 10 seconds 
// (but it depends on the RNG initialization).
// see end of file for description of cmd line args

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>

#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "custom/c_config.h"
#include "custom/c_calc_function.cu"

struct cinstance {
	int match;
	int mdim;
	int* rules;
	int rules_len;
	float* indv;
	float* rat;
};

template<int mnum, int mdim, int mcond>
__global__ void rating_kernel(struct cinstance inst)
{
	const int rlen = inst.rules_len;
	const int* const rrules = inst.rules;

	/* mutation */
	if(tx == 0 && ty == 0) {
		rend = srules + rlen - 1;
		res = sind + mnum * mdim * mdim;
	}

	/* caching of rules to speed up access */
	for(int i = RIDX(ty, tx); i < rlen; i += mdim*mdim)
		srules[i] = rrules[i];

	const float* const indv = inst.indv;
	const int iwidth = mnum*mdim*mdim;
	for(int i = RIDX(ty, tx); i < iwidth; i += mdim*mdim) {
		sind[i] = indv[i];
	}
	__syncthreads();

	c_calc_res<mdim, mcond>(inst.match, 1.f);
	if(tx == 0 && ty == 0)
		*inst.rat = shrd_rating;
}

float penalty(struct cinstance& i, float* indv)
{
	const size_t space =(2 * i.mdim * i.mdim + i.mdim * i.mdim) * sizeof(*indv);
	const dim3 blocks(BLOCKS);
	const dim3 threads(i.mdim, i.mdim);

	CUDA_CALL(hipMemcpy(i.indv, indv, 2 * i.mdim * i.mdim * sizeof(*indv), hipMemcpyHostToDevice));

	switch(i.mdim) {
	case 5:
		rating_kernel<2, 5, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 6:
		rating_kernel<2, 6, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 7:
		rating_kernel<2, 7, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 8:
		rating_kernel<2, 8, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 9:
		rating_kernel<2, 9, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 10:
		rating_kernel<2, 10, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 11:
		rating_kernel<2, 11, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 12:
		rating_kernel<2, 12, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 13:
		rating_kernel<2, 13, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 14:
		rating_kernel<2, 14, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 15:
		rating_kernel<2, 15, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	case 16:
		rating_kernel<2, 16, COND_UPPER_RIGHT><<<blocks, threads, space>>>(i);
		CUDA_CALL(hipGetLastError());
		break;
	}

	float rat;
	CUDA_CALL(hipMemcpy(&rat, i.rat, sizeof(rat), hipMemcpyDeviceToHost));
	return rat;
}


float *
make(int rows, int cols)
{
	return (float *) malloc(rows * cols * sizeof(float));
}

void
copy(int rows, int cols, const float * src, float * dest)
{
	memcpy(dest, src, rows * cols * sizeof(float));
}

void
show(FILE * f, int mcount, int mwidth, const float * mat)
{
	for (int c = 0; c < mcount; c++) {
		for (int j = 0; j < mwidth; j++) {
			fprintf(f, "----");
		}
		fprintf(f, "letter %d\n", c);
		for (int i = 0; i < mwidth; i++) {
			for (int j = 0; j < mwidth; j++) {
				fprintf(
						f,
						"%4.2f ",
						mat[c * mwidth * mwidth
								+ i * mwidth + j]);
			}
			fprintf(f, "\n");
		}
	}
}

void
plus(int rows, int cols, const float * a, const float * b, float * c)
{
	for (int i = 0; i < rows * cols; i++) {
		c[i] = a[i] + b[i];
	}
}

void
times(int rows, int mid, int cols, const float * a, const float * b, float * c)
{
	for (int i = 0; i < rows; i++) {
		for (int k = 0; k < cols; k++) {
			int s = 0;
			for (int j = 0; j < mid; j++) {
				s += a[i * mid + j] * b[j * cols + k];
			}
			c[i * cols + k] = s;
		}
	}
}

// write individuumpretation of word w
// into pre-allocated result matrix
void
eval(int * rule, int mwidth, float * individuum, float * result)
{
	assert(rule[0] >= 0);
	copy(mwidth, mwidth, individuum + mwidth * mwidth * rule[0], result);
	for (int i = 1; rule[i] >= 0; i++) {
		float * accu = make(mwidth, mwidth);
		times(mwidth, mwidth, mwidth, result,
				individuum + mwidth * mwidth * rule[i], accu);
		copy(mwidth, mwidth, accu, result);
		free(accu);
	}
}

float
penalty(int * lhs, int * rhs,
// lhs, rhs are strings 0,1,2,..,
// terminated by negative number.
// FIXME: empty string not handled correctly
		int mwidth, float * individuum
		// individuumpretation  is array of matrices
		)
{
	float * l = make(mwidth, mwidth);
	eval(lhs, mwidth, individuum, l);
	float * r = make(mwidth, mwidth);
	eval(rhs, mwidth, individuum, r);
	float s = 0;
	for (int row = 0; row < mwidth; row++) {
		for (int col = 0; col < mwidth; col++) {
			float x = l[row * mwidth + col];
			float y = r[row * mwidth + col];
			float p = 0;
			if ((row == 0) && (col == mwidth - 1)) {
				p = (x > y) ? 0 : 10000 * (y - x + 1);
			} else {
				p = (x >= y) ? 0 : y * y - x * x;
			}
			if (p > 1000000)
				p = 1000000;
			s += p;
		}
	}
	free(l);
	free(r);
	return s;
}

// ensure special shape: 
// first column is (1,0..0)^T, last row is (0..0,1)
void
patch(int mwidth, float * m)
{
	for (int i = 0; i < mwidth; i++) {
		m[i * mwidth + 0] = 0;
		m[(mwidth - 1) * mwidth + i] = 0;
	}
	m[0 * mwidth + 0] = 1;
	m[(mwidth - 1) * mwidth + mwidth - 1] = 1;
}

// fill randomly with 0,1
void
fill(int mwidth, float * m)
{
	for (int i = 0; i < mwidth; i++) {
		for (int j = 0; j < mwidth; j++) {
			m[i * mwidth + j] = random() % 2;
		}
	}
	patch(mwidth, m);
}

// change one position (downwards only)
void
mutate(int mcount, int mwidth, float * individuum)
{
	int letter = random() % mcount;
	int row = random() % (mwidth - 1);
	int col = 1 + random() % (mwidth - 1);
	float * pos = individuum + letter * mwidth * mwidth + row * mwidth + col;
	int newP = *pos - 1;
	if (newP < 0)
		newP = 0;
	*pos = newP;
}

// this is the main trick, stolen from Dieter Hofbauer, 
// who had this in MultumNonMulta already in 2006:

// increase weights on path that corresponds to some
// error (= weight increase) in some position.

// find an index pair (p,q) such that lhs[p,q] < rhs[p,q],
// then find a random (!) path (sequence of indices)
// p = p_0 , p_1, p_2, .. , p_n = q,
// then for each i, increase the value of 
// individuum[p_i, p_i+1] in the individuumpretation of
// letter  lhs[i].

void
path_mutate(int * lhs, int * rhs, int mcount, int mwidth, float * individuum)
{
	float * lres = make(mwidth, mwidth);
	float * rres = make(mwidth, mwidth);

	eval(lhs, mwidth, individuum, lres);
	eval(rhs, mwidth, individuum, rres);

	int top = 0;
	int * rows = (int*)malloc(mwidth * mwidth * sizeof(float));
	int * cols = (int*)malloc(mwidth * mwidth * sizeof(float));

	for (int row = 0; row < mwidth; row++) {
		for (int col = 0; col < mwidth; col++) {

			int special = (row == 0) && (col == mwidth - 1);
			int l = lres[row * mwidth + col];
			int r = rres[row * mwidth + col];
			int ok = special ? (l > r) : (l >= r);

			if (!ok) {
				rows[top] = row;
				cols[top] = col;
				top++;
			}
		}
	}

	if (0 == top) {
		fprintf(stdout, "what");
		show(stdout, mcount, mwidth, individuum);
		// exit (0);
		return;
	}

	int i = random() % top;
	int l = rows[i];
	int r = cols[i];

	free(rows);
	free(cols);
	free(lres);
	free(rres);

	// now we have the position of the error in (l,q)

	for (int i = 0; lhs[i] >= 0; i++) {
		int c = lhs[i];
		int goal = lhs[i + 1] < 0 ? r : 1 + random() % (mwidth - 2);
		float * pos = individuum + c * mwidth * mwidth + l * mwidth
				+ goal;
		int newP = *pos + 1;
		if (newP < 1)
			newP = 1;
		*pos = newP;
		l = goal;
	}

}

// try to overwrite this individuum with a better one
int
anneal(cinstance& inst, int total, int * lhs, int * rhs, int mcount, int mwidth,
		float * individuum)
{
//	int best = penalty(lhs, rhs, mwidth, individuum);
	float best = penalty(inst, individuum);
	if (0 == best)
		return 0;
	// fprintf (stdout, "anneal start %d\n", best);

	size_t s = mcount * mwidth * mwidth * sizeof(float);
	float * candidate = (float*)malloc(s);

	for (int steps = 0; steps < total; steps++) {
		memcpy(candidate, individuum, s);
		mutate(mcount, mwidth, candidate);
//		int p = penalty(lhs, rhs, mwidth, candidate);
		float p = penalty(inst, individuum);
		int luck = 0 == random() % total;
		if (p <= best || luck) {
			memcpy(individuum, candidate, s);
			best = p;
		}
	}
	// fprintf (stdout, "anneal end %d\n", best);

	free(candidate);
	return best;
}

int
bits(int x)
{
	int c = 0;
	while (x > 0) {
		x >>= 1;
		c++;
	}
	return c;
}

void
census(char * s, int size, int * data)
{
	int maxbits = 32;
	int * tab = (int*)malloc(maxbits * sizeof(int));
	for (int i = 0; i < maxbits; i++) {
		tab[i] = 0;
	}
	for (int i = 0; i < size; i++) {
		tab[bits(data[i])]++;
	}
	printf("census: number of items with %s of given bit width\n", s);
	for (int i = 0; i < maxbits; i++) {
		if (tab[i] > 0) {
			printf("%d: %d, ", i, tab[i]);
		}
	}
	printf("\n");
	free(tab);
}

void
census(char * s, int size, float * data)
{
	int maxbits = 32;
	int * tab = (int*)malloc(maxbits * sizeof(int));
	for (int i = 0; i < maxbits; i++) {
		tab[i] = 0;
	}
	for (int i = 0; i < size; i++) {
		tab[bits((int)data[i])]++;
	}
	printf("census: number of items with %s of given bit width\n", s);
	for (int i = 0; i < maxbits; i++) {
		if (tab[i] > 0) {
			printf("%d: %d, ", i, tab[i]);
		}
	}
	printf("\n");
	free(tab);
}

static inline int timespec_subtract(struct timespec *result,
                                    struct timespec *after,
                                    struct timespec *before)
{
        result->tv_nsec = after->tv_nsec - before->tv_nsec;

        if (result->tv_nsec < 0) {
                /* Borrow 1sec from 'tv_sec' if subtraction -ve */
                result->tv_nsec += 1000000000;
                result->tv_sec = after->tv_sec - before->tv_sec - 1;

                return 1;
        } else {
                result->tv_sec = after->tv_sec - before->tv_sec;
                return 0;
        }
}

void
evolution(cinstance& inst, int size, int asteps, int * lhs, int * rhs, int mcount, int mwidth)
{
	int s = mcount * mwidth * mwidth;
	float * pop = (float*)malloc(size * s * sizeof(float));
	float * pen = (float*)malloc(size * sizeof(float));
	int * age = (int*)malloc(size * sizeof(int));

	for (int p = 0; p < size; p++) {
		float * individuum = pop + p * mcount * mwidth * mwidth;

		for (int c = 0; c < mcount; c++) {
			fill(mwidth, individuum + c * mwidth * mwidth);
		}

//		pen[p] = penalty(lhs, rhs, mwidth, individuum);
		pen[p] = penalty(inst, individuum);
		age[p] = 0;
	}

	int globally_best = pen[0];

	struct timespec before, after, elapsed;

	clock_gettime(CLOCK_MONOTONIC, &before);
	for (int step = 0;; step++) {
		int parent = random() % size;
		float * individuum = (float*)malloc(s * sizeof(float));
		memcpy(individuum, pop + parent * s, s * sizeof(int));

		path_mutate(lhs, rhs, mcount, mwidth, individuum);
		int best = anneal(inst, asteps, lhs, rhs, mcount, mwidth, individuum);

		int child = random() % size;

		if (best < pen[child]) {
			if (best < globally_best) {
				fprintf(
						stdout,
						"step %5d: parent from %d with fit %d age %d replaces child at %d with fit %d age %d\n",
						step, parent, best, age[parent],
						child, pen[child], age[child]);
				globally_best = best;
			}

			memcpy(pop + child * s, individuum, s * sizeof(int));
			pen[child] = best;
			age[child] = age[parent] + 1;
		}

		if (0 == best) {
			show(stdout, mcount, mwidth, individuum);
			break;
		}

		if (0 == step % 1000) {
			clock_gettime(CLOCK_MONOTONIC, &after);
			timespec_subtract(&elapsed, &after, &before);
			printf("step %d need %ld.%03lds\n", step, elapsed.tv_sec, elapsed.tv_nsec / (long int)1e6);
			census("penalty", size, pen);
			census("age", size, age);
			clock_gettime(CLOCK_MONOTONIC, &before);
		}
		free(individuum);
	}
	free(age);
	free(pop);
}

void
init_random_generator()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	srandom(tv.tv_usec);
}

int
main(int argc, char ** argv)
{
	if (4 != argc) {
		fprintf(stderr, "cmd line arguments:\n");
		fprintf(stderr, "  int mwidthension of matrices,\n");
		fprintf(stderr, "  int size of population,\n");
		fprintf(stderr, "  int number of annealing steps.\n");
		fprintf(stderr, "example: ./matrix 5 100 100\n");
		exit(-1);
	}

	int mwidth;
	sscanf(argv[1], "%d", &mwidth);
	int pop;
	sscanf(argv[2], "%d", &pop);
	int ann;
	sscanf(argv[3], "%d", &ann);

	int mcount = 2;

	int lhs[] = { 0, 0, 1, 1, -1 };
	int rhs[] = { 1, 1, 1, 0, 0, 0, -1 };

	const int rlen = 13;
	int rules[] = {-1, 0, 0, 1, 1, -1, 1, 1, 1, 0, 0, 0, -1};

	struct cinstance inst;
	inst.mdim  = mwidth;
	inst.match = MATCH_ALL;
	inst.rules_len = rlen;
	CUDA_CALL(hipMalloc(&(inst.rat), sizeof(*inst.rat)));
	CUDA_CALL(hipMalloc(&(inst.rules), rlen * sizeof(*rules)));
	CUDA_CALL(hipMalloc(&(inst.indv), 2*mwidth*mwidth*sizeof(*inst.indv)));
	CUDA_CALL(hipMemcpy(inst.rules, rules, rlen * sizeof(*rules), hipMemcpyHostToDevice));

	// int lhs [] = { 0,0, -1};
	// int rhs [] = { 0,1,0, -1};

	init_random_generator();
	evolution(inst, pop, ann, lhs, rhs, mcount, mwidth);

	hipFree(inst.rat);
	hipFree(inst.rules);
	hipFree(inst.indv);
}
