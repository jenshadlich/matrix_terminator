#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>

#include "config.h"
#include "instance.h"
#include "evo.h"

#include "matrix_print.h"
#include "matrix_copy.h"

/*
 * Allocate memory for the parent matrices. the memory is layouted for faster
 * access. The block count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_parent_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents    * /* there are n parents per block */
		    inst->width_per_inst *
		    sizeof(float);

	inst->dev_parent_ext = make_hipExtent(width,
					       inst->dim.matrix_height,
					       inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_parent_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_parent_ext));

	inst->dev_parent = pitched_ptr;
}

/*
 * Allocate memory for the child matrices. the memory is layouted for faster
 * access. The bloc count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_child_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents * inst->dim.childs * /* each parent should have n childs */
		    inst->width_per_inst * sizeof(float);

	inst->dev_child_ext = make_hipExtent(width,
					      inst->dim.matrix_height,
					      inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_child_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_child_ext));
	inst->dev_child = pitched_ptr;
}

/**
 * Allocate the matrix for each thread which is 
 * needed for the multiplication and evaluation.
 */
void alloc_result_matrix(struct instance *inst)
{
	inst->dev_res_ext = make_hipExtent(inst->dim.childs * inst->dim.parents *
					    2 * inst->dim.matrix_width * sizeof(float),
					    inst->dim.matrix_height,
					    inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_res_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_res_ext));
	inst->dev_res = pitched_ptr;
}

inline int get_evo_threads(struct instance *inst) {
	return inst->dim.parents * inst->dim.childs;
}

void alloc_rating(struct instance *inst)
{
	inst->dev_crat_ext = make_hipExtent(2 * get_evo_threads(inst) * sizeof(float),
	 			    	     1,
	 			    	     inst->dim.blocks);

	inst->dev_prat_ext = make_hipExtent(inst->dim.parents * sizeof(float),
					     1,
					     inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_crat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_crat_ext));
	inst->dev_crat = pitched_ptr;

	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_prat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_prat_ext));
	inst->dev_prat = pitched_ptr;
}

void init_rnd_generator(struct instance *inst, int seed)
{	
	hiprandState *rnd_states;
	const int count = get_evo_threads(inst);

	CUDA_CALL(hipMalloc((void **)&rnd_states, 
			     count * BLOCKS * sizeof(hiprandState)));
	setup_rnd_kernel<<<BLOCKS, count>>>(rnd_states, seed);
	CUDA_CALL(hipGetLastError());
	inst->rnd_states = rnd_states;
}

void set_num_matrices(struct instance* inst)
{
	int m = INT_MIN;
	for(int i = 0; i < inst->rules_len; i++)
		m = max(m, inst->rules[i]);

	inst->num_matrices = m + 1; /* matrices are zero based */
	printf("num_matrices set to %d\n", inst->num_matrices);
}

void init_instance(struct instance* inst)
{
	inst->rule_count = 3;
	inst->rules_len  = 22;
	inst->rules = (int*)malloc(sizeof(int) * inst->rules_len);
	inst->rules[0] = MUL_SEP;
	inst->rules[1] = 1;
	inst->rules[2] = 1;
	inst->rules[3] = 1;
	inst->rules[4] = MUL_SEP;
	inst->rules[5] = 0;
	inst->rules[6] = MUL_SEP;

	inst->rules[7] = 0;
	inst->rules[8] = 0;
	inst->rules[9] = MUL_SEP;
	inst->rules[10] = 0;
	inst->rules[11] = 1;
	inst->rules[12] = 0;
	inst->rules[13] = MUL_SEP;

	inst->rules[14] = 0;
	inst->rules[15] = 0;
	inst->rules[16] = 0;
	inst->rules[17] = MUL_SEP;
	inst->rules[18] = 1;
	inst->rules[19] = 0;
	inst->rules[20] = 0;
	inst->rules[21] = MUL_SEP;

	inst->delta = 0.1;
	inst->match = MATCH_ALL;
	inst->cond_left  = COND_UPPER_RIGHT;
	inst->cond_right = COND_UPPER_RIGHT;

	inst->dim.blocks  = BLOCKS;
	inst->dim.childs  = CHILDS;
	inst->dim.parents = PARENTS;
	inst->dim.matrix_width  = MATRIX_WIDTH;
	inst->dim.matrix_height = MATRIX_HEIGHT;
	
	inst->rounds = 0;

	set_num_matrices(inst);

	inst->width_per_inst = inst->num_matrices *    /* there are n matrices needed for the rules */
			       inst->dim.matrix_width; /* each one has a fixed width */

	alloc_parent_matrix(inst);
	alloc_child_matrix(inst);
	alloc_result_matrix(inst);
	alloc_rating(inst);
	init_rnd_generator(inst, time(0));
}

void cleanup(struct instance *inst, struct instance * dev_inst) {
	free(inst->rules);

	hipFree(dev_inst);
	/* dev_inst-> rules? */

	hipFree(inst->rnd_states);
	hipFree(inst->dev_child.ptr);
	hipFree(inst->dev_parent.ptr);
	hipFree(inst->dev_res.ptr);
	hipFree(inst->dev_crat.ptr);
	hipFree(inst->dev_prat.ptr);
}

struct instance* create_dev_inst(struct instance *inst)
{
	struct instance *dev_inst;
	int *rules = inst->rules;
	CUDA_CALL(hipMalloc(&(inst->rules), inst->rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(inst->rules, rules, inst->rules_len * sizeof(int),
					hipMemcpyHostToDevice));
	CUDA_CALL(hipMalloc(&dev_inst, sizeof(*dev_inst)));
	CUDA_CALL(hipMemcpy(dev_inst, inst, sizeof(*dev_inst),
					hipMemcpyHostToDevice));

	inst->rules = rules;
	return dev_inst;
}

void copy_inst_dev_to_host(struct instance *dev, struct instance *host)
{
	int *rules = host->rules;
	CUDA_CALL(hipMemcpy(host, dev, sizeof(*dev), hipMemcpyDeviceToHost));
	host->rules = rules;
}

int main(int argc, char** argv)
{
	struct instance inst;
	struct instance *dev_inst;

	init_instance(&inst);
	dev_inst = create_dev_inst(&inst);

	setup_parent_kernel<<<BLOCKS, inst.dim.matrix_height>>>(dev_inst);
	CUDA_CALL(hipGetLastError());
	//print_parent_matrix(&inst);

	int evo_threads = get_evo_threads(&inst);
	evo_kernel<<<BLOCKS, evo_threads>>>(dev_inst);
	CUDA_CALL(hipGetLastError());

	copy_inst_dev_to_host(dev_inst, &inst);

	printf("Needed rounds: %d\n", inst.rounds);
	printf("Result is block: %d, parent: %d\n", inst.res_block, inst.res_parent);
	print_parent_matrix(&inst, inst.res_block, inst.res_parent);

	printf("Clean up and exit.\n");
	cleanup(&inst, dev_inst);
}
