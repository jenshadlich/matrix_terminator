#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>

#include "instance.h"

#include "evo.h"
#include "evo_rating.h"
#include "evo_setup.h"

#include "matrix_print.h"
#include "matrix_copy.h"

/*
 * Allocate memory for the parent matrices. the memory is layouted for faster
 * access. The block count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_parent_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents    * /* there are n parents per block */
		    inst->width_per_inst *
		    sizeof(double);

	inst->dev_parent_ext = make_hipExtent(width,
					       inst->dim.matrix_height,
					       inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_parent_ext));

	inst->dev_parent = pitched_ptr;
}

/*
 * Allocate memory for the child matrices. the memory is layouted for faster
 * access. The bloc count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_child_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents * inst->dim.childs * /* each parent should have n childs */
		    inst->width_per_inst * sizeof(double);

	inst->dev_child_ext = make_hipExtent(width,
					      inst->dim.matrix_height,
					      inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_child_ext));
	inst->dev_child = pitched_ptr;
}

/**
 * Allocate the matrix for each thread which is 
 * needed for the multiplication and evaluation.
 */
void alloc_result_matrix(struct instance *inst)
{
	inst->dev_res_ext = make_hipExtent(inst->dim.childs * inst->dim.parents *
					    2 * inst->dim.matrix_width * sizeof(double),
					    inst->dim.matrix_height,
					    inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_res_ext));
	inst->dev_res = pitched_ptr;
}

void alloc_sparam(struct instance *inst)
{
	inst->dev_sparam_ext = make_hipExtent(inst->dim.childs * inst->dim.parents * sizeof(double),
					       1,
					       inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_sparam_ext));
	inst->dev_sparam = pitched_ptr;
}
inline int get_evo_threads(struct instance *inst) {
	return inst->dim.parents * inst->dim.childs;
}

void alloc_rating(struct instance *inst)
{
	inst->dev_crat_ext = make_hipExtent(2 * get_evo_threads(inst) * sizeof(double),
	 			    	     1,
	 			    	     inst->dim.blocks);

	inst->dev_prat_ext = make_hipExtent(inst->dim.parents * sizeof(double),
					     1,
					     inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_crat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_crat_ext));
	inst->dev_crat = pitched_ptr;

	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_prat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 0, inst->dev_prat_ext));
	inst->dev_prat = pitched_ptr;
}

void init_rnd_generator(struct instance *inst, int seed)
{	
	hiprandState *rnd_states;
	const int count = max(get_evo_threads(inst), MATRIX_HEIGHT);

	CUDA_CALL(hipMalloc((void **)&rnd_states, 
			     count * BLOCKS * sizeof(hiprandState)));
	setup_rnd_kernel<<<BLOCKS, count>>>(rnd_states, seed);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	inst->rnd_states = rnd_states;
}

void set_num_matrices(struct instance* inst)
{
	int m = INT_MIN;
	for(int i = 0; i < inst->rules_len; i++)
		m = max(m, inst->rules[i]);

	inst->num_matrices = m + 1; /* matrices are zero based */
	printf("num_matrices set to %d\n", inst->num_matrices);
}

void init_instance(struct instance* inst, char* rules)
{
	inst->rules_len  = strlen(rules);
	inst->rules = (int*)malloc(sizeof(int) * inst->rules_len);

	for(int i = 0; i < inst->rules_len; i++) {
		if(rules[i] > 96)
			inst->rules[i] = (rules[i] == 'X') ? MUL_SEP : rules[i] - 'a';
		else
			inst->rules[i] = (rules[i] == 'X') ? MUL_SEP : rules[i] - '0';
	}

	inst->delta = 1;
	inst->match = MATCH_ALL;
	inst->cond_left  = COND_UPPER_LEFT;
	inst->cond_right = COND_UPPER_LEFT;

	inst->dim.blocks  = BLOCKS;
	inst->dim.childs  = CHILDS;
	inst->dim.parents = PARENTS;
	inst->dim.matrix_width  = MATRIX_WIDTH;
	inst->dim.matrix_height = MATRIX_HEIGHT;
	
	inst->rounds = 0;
	inst->isnan = 0;

	inst->res_block = 0;
	inst->res_parent = 0;
	inst->res_child_block = 0;
	inst->res_child_idx = 0;

	set_num_matrices(inst);

	inst->width_per_inst = inst->num_matrices *    /* there are n matrices needed for the rules */
			       inst->dim.matrix_width; /* each one has a fixed width */

	alloc_parent_matrix(inst);
	alloc_child_matrix(inst);
	alloc_result_matrix(inst);
	alloc_rating(inst);
	alloc_sparam(inst);
	init_rnd_generator(inst, time(0));
}

void cleanup(struct instance *inst, struct instance * dev_inst) {
	free(inst->rules);

	hipFree(dev_inst);
	/* dev_inst-> rules? */

	hipFree(inst->rnd_states);
	hipFree(inst->dev_child.ptr);
	hipFree(inst->dev_parent.ptr);
	hipFree(inst->dev_res.ptr);
	hipFree(inst->dev_crat.ptr);
	hipFree(inst->dev_prat.ptr);
	hipFree(inst->dev_sparam.ptr);
}

struct instance* create_dev_inst(struct instance *inst)
{
	struct instance *dev_inst;
	int *rules = inst->rules;
	int *dev_rules;
	CUDA_CALL(hipMalloc(&dev_rules, inst->rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(dev_rules, rules, inst->rules_len * sizeof(int),
					hipMemcpyHostToDevice));
	inst->rules = dev_rules;
	CUDA_CALL(hipMalloc(&dev_inst, sizeof(*dev_inst)));
	CUDA_CALL(hipMemcpy(dev_inst, inst, sizeof(*dev_inst),
					hipMemcpyHostToDevice));

	inst->rules = rules;
	return dev_inst;
}

void copy_inst_dev_to_host(struct instance *dev, struct instance *host)
{
	int *rules = host->rules;
	CUDA_CALL(hipMemcpy(host, dev, sizeof(*dev), hipMemcpyDeviceToHost));
	host->rules = rules;
}

int main(int argc, char** argv)
{
	if(argc < 2) {
		printf("Please supply a rule\n");
		exit(1);
	}

	/* there is no runtime limit for kernels */
	CUDA_CALL(hipSetDevice(0));

	struct instance inst;
	struct instance *dev_inst;

	init_instance(&inst, argv[1]);
	dev_inst = create_dev_inst(&inst);

	printf("Rules: ");
	print_rules(&inst);

	setup_parent_kernel<<<BLOCKS, inst.dim.matrix_height>>>(dev_inst);
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	int evo_threads = get_evo_threads(&inst);
	dim3 blocks(BLOCKS, PARENTS*CHILDS);
	dim3 threads(MATRIX_WIDTH, MATRIX_HEIGHT);

	setup_sparam<<<BLOCKS, evo_threads>>>(dev_inst);
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	// Prepare
	hipEvent_t start, stop;
	float elapsedTime;
	float elapsedTimeTotal = 0.f;

	int width = inst.dim.parents * inst.dim.blocks;
	double *rating = (double*)malloc(width * sizeof(double));
	int rounds = -1;

	int max_rounds = 500;
	int block = 0; int thread = 0;

	for(int i = 0; i < max_rounds; i++) {
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// Start record
		hipEventRecord(start, 0);

		evo_kernel<<<BLOCKS, evo_threads>>>(dev_inst, 0);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		evo_calc_res<<<blocks, threads>>>(dev_inst);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		evo_kernel<<<BLOCKS, evo_threads>>>(dev_inst, 1);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
		elapsedTimeTotal += elapsedTime;
		// Clean up:
		hipEventDestroy(start);
		hipEventDestroy(stop);

		if(i & 1111 != 0 && i != (max_rounds - 1))
			continue;

//		print_parent_matrix_pretty(&inst, inst.res_block, inst.res_parent);
//		print_parent_ratings(&inst);
//		print_sparam(&inst);
		copy_parent_rating_dev_to_host(&inst, rating);
		for(int j = 0; j < width; j += PARENTS) {
			if(rating[j] == 0.) {
				block = j / PARENTS;
				thread = j % PARENTS;
				rounds = i;
				i = max_rounds;
				break;
			}
		}
	}

	free(rating);
	copy_inst_dev_to_host(dev_inst, &inst);

	print_sparam(&inst);
	print_parent_ratings(&inst);
	printf("Result:\n");
	print_result_matrix_pretty(&inst, block, thread);
	printf("Parents:\n");
	print_parent_matrix_pretty(&inst, block, thread);
	print_rules(&inst);
	printf("Time needed: %f\n", elapsedTimeTotal);
	printf("Needed rounds: %d\n", rounds);
	printf("Is NaN: %d\n", inst.isnan);
	printf("Result is block: %d, parent: %d\n", block, thread);
	printf("Result was in block: %d, child: %d\n", inst.res_child_block, inst.res_child_idx);

	printf("Clean up and exit.\n");
	cleanup(&inst, dev_inst);

	if(rounds == -1)
		return 0;

	return 1;
}
