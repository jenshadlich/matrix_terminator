/*
 * matrix_generator.c
 *
 *  Created on: Sep 22, 2011
 *      Author: tkalbitz
 */

extern "C"
{
#include "matrix_generator.h"
}

#include "evo_info.h"
#include "evo_error.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <ctype.h>
#include <errno.h>

#include <sys/wait.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "evo.h"
#include "evo_rating.h"
#include "evo_setup.h"

#include "matrix_print.h"
#include "matrix_copy.h"
#include "ya_malloc.h"
#include "plot_log.h"

static void copy_result_to_buffer(struct instance* inst,
				  int block, int parent,
				  double* buffer)
{
	int width = inst->dim.parents    * /* there are n parents per block */
		    inst->width_per_inst *
		    sizeof(double) *
		    inst->dim.matrix_height * inst->dim.blocks;

	double* parent_cpy = (double*)ya_malloc(width);
	memset(parent_cpy, 1, width);
	copy_parents_dev_to_host(inst, parent_cpy);

	int line = inst->dim.parents *  inst->width_per_inst;
	int block_offset = line * inst->dim.matrix_height;
	double* block_ptr = parent_cpy + block_offset * block;

	double* to;
	double* from;

	for(int i = 0; i < inst->dim.matrix_height; i++) {
		to   = buffer     + i * inst->width_per_inst;
		from = block_ptr + (i * line);
		memcpy(to, from, inst->width_per_inst * sizeof(double));
	}

	free(parent_cpy);
}

int evo_run(const int     instance,
	    const int     cycles,
	    double* const result)
{
	struct evo_info_t* const evo_info = evo_get(instance);
	if(evo_info == NULL)
		return E_INVALID_INST;

	struct instance *inst = evo_info->inst;
	struct instance *dev_inst;
	int *dev_rules;

	dev_inst = inst_create_dev_inst(inst, &dev_rules);
	int evo_threads = get_evo_threads(inst);

	const dim3 blocks(BLOCKS, PARENTS*CHILDS);
	const dim3 threads(inst->dim.matrix_width, inst->dim.matrix_height);
	const dim3 copy_threads(inst->dim.matrix_width, inst->dim.matrix_height);
	const dim3 setup_threads(inst->dim.matrix_width * inst->dim.matrix_height);

	setup_childs_kernel<<<BLOCKS, setup_threads>>>(dev_inst, false);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	setup_sparam<<<BLOCKS, evo_threads>>>(dev_inst,
			evo_info->sparam, evo_info->mut_rate,
			evo_info->recomb_rate, false);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	// Prepare
	hipEvent_t start, stop;
	float elapsedTime;
	float elapsedTimeTotal = 0.f;

	const int width = inst->dim.parents * inst->dim.blocks;
	double * const rating = (double*)ya_malloc(width * sizeof(double));
	int rounds = INT_MAX;
	int block = 0; int thread = 0;

	evo_calc_res<<<blocks, threads>>>(dev_inst);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	evo_kernel_part_two<<<BLOCKS, copy_threads>>>(dev_inst);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	CUDA_CALL(hipGetLastError());

	for(int i = 0; i < cycles; i++) {
		if(i % 300 == 0) {
			setup_childs_kernel<<<BLOCKS, setup_threads>>>(dev_inst, true);
			CUDA_CALL(hipGetLastError());
			evo_calc_res<<<blocks, threads>>>(dev_inst);
			CUDA_CALL(hipGetLastError());
			evo_kernel_part_two<<<BLOCKS, copy_threads>>>(dev_inst);
			CUDA_CALL(hipGetLastError());
			setup_sparam<<<BLOCKS, evo_threads>>>(dev_inst,
					evo_info->sparam,
					evo_info->mut_rate,
					evo_info->recomb_rate, true);
			CUDA_CALL(hipGetLastError());
		}

		hipEventCreate(&start);
		hipEventCreate(&stop);
		// Start record
		hipEventRecord(start, 0);

		evo_kernel_part_one<<<BLOCKS, evo_threads>>>(dev_inst);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		evo_calc_res<<<blocks, threads>>>(dev_inst);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		evo_kernel_part_two<<<BLOCKS, copy_threads>>>(dev_inst);
		CUDA_CALL(hipGetLastError());
		hipDeviceSynchronize();
		CUDA_CALL(hipGetLastError());

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
		elapsedTimeTotal += elapsedTime;
		// Clean up:
		hipEventDestroy(start);
		hipEventDestroy(stop);

		copy_parent_rating_dev_to_host(inst, rating);

		for(int j = 0; j < width; j += PARENTS) {
			if(rating[j] == 0.) {
				block = j / PARENTS;
				thread = j % PARENTS;
				rounds = i;
				i = cycles;
				break;
			}
		}
	}

	free(rating);
	copy_result_to_buffer(inst, block, thread, result);

	hipFree(dev_inst);
	hipFree(dev_rules);
	return rounds;
}

