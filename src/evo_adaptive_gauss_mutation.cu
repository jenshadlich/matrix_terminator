#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

__device__ static double evo_mut_new_value(struct instance * const inst,
					   hiprandState     * const rnd_state)
{
	/* we want to begin with small numbers */
//	const int tmp = (int)inst->parent_max;
	const int tmp = (inst->parent_max > 10) ? 10 : (int)inst->parent_max;

	const int rnd_val = (hiprand(rnd_state) % (tmp - 1)) + 1;
	int factor = (int)(rnd_val / inst->delta);
	if((factor * inst->delta) < 1.0)
		factor++;

	if(factor * inst->delta < 1.)
		return 1;

	return factor * inst->delta;
}

__device__ void evo_ensure_constraints(struct instance * const inst,
				       struct memory   * const mem,
				       hiprandState     * const rnd_state)
{
	double* const row   = C_ROW(0);
	double* const lrow  = C_ROW(inst->dim.matrix_height-1);

	const int end = mem->c_end;

	for(int start = mem->c_zero; start < end; start += inst->dim.matrix_width) {
		const int lidx = start + inst->dim.matrix_width - 1;

		if(inst->cond_left == COND_UPPER_LEFT && row[start] < 1.0) {
			row[start] = evo_mut_new_value(inst, rnd_state);
		} else if(inst->cond_left == COND_UPPER_RIGHT && row[lidx] < 1.0) {
			row[lidx] = evo_mut_new_value(inst, rnd_state);
		} else if(inst->cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
			if(row[start] < 1.0)
				row[start] = evo_mut_new_value(inst, rnd_state);

			if(lrow[lidx] < 1.0)
				lrow[lidx] = evo_mut_new_value(inst, rnd_state);
		} else {
			/*
			 * This should be recognized ;) It's only a 1.3 card
			 *  so there is no printf :/
			 */
			for(int i = 0; i < inst->dim.matrix_width; i++) {
				row[start + i] = 1337;
				lrow[start + i] = 1337;
			}
		}
	}
}

__device__ void evo_mutation(struct instance * const inst,
			     struct memory   * const mem,
			     hiprandState     * const rnd_s)
{
	const int rows = inst->dim.matrix_height;
	const double delta = inst->delta;
	const uint32_t elems = inst->dim.matrix_width*inst->dim.matrix_height*inst->num_matrices;
	double tmp;

//	SP(tx) = SP(tx) * exp(hiprand_normal(rnd_s) /
//			sqrtf(inst->num_matrices * inst->dim.matrix_height));
	SP(tx) *= exp( (1 / sqrtf(inst->num_matrices * inst->dim.matrix_height * inst->dim.matrix_height)) * hiprand_normal(rnd_s));
	SP(tx) = min(max(SP(tx), 2*delta), inst->parent_max);

	MR(tx) = MR(tx) + (hiprand_normal(rnd_s) / 20);
	MR(tx) = min(max(MR(tx), 1./elems), 1.);

	const double mr = MR(tx);
	const double sp = SP(tx);

	#pragma unroll
	for(int r = 0; r < rows; r++) {
		double* const row = C_ROW(r);

		for(int c = mem->c_zero; c < mem->c_end; c++) {

			if(hiprand_uniform(rnd_s) > mr) {
				if(hiprand_uniform(rnd_s) < mr/10) {
					row[c] = 0.;
				} if(hiprand_uniform(rnd_s) < mr/10) {
					row[c] = evo_mut_new_value(inst, rnd_s);
				}
				continue;
			}

			tmp = (double)(hiprand_normal(rnd_s) * sp);
			tmp = (tmp < 0 ? -1 : 1) * max(delta, fabs(tmp));
			tmp = row[c] + tmp;
			/* we want x * delta, where x is an int */
			tmp = ((unsigned long)(tmp / delta)) * delta;
			tmp = max(tmp, 0.0);
			tmp = min(inst->parent_max, tmp);

			row[c] = tmp;
		}
	}

	evo_ensure_constraints(inst, mem, rnd_s);
}
