#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "instance.h"

#include "evo_rating.h"
#include "evo_memory.cu"

__shared__ double res[2][MATRIX_HEIGHT][MATRIX_WIDTH];

__device__ inline void eval_set_res_matrix_to_zero()
{
	res[0][threadIdx.y][threadIdx.x] = 0.;
	res[1][threadIdx.y][threadIdx.x] = 0.;
}

__device__ inline void eval_set_res_matrix_to_identity()
{
	if(threadIdx.x != threadIdx.y) {
		res[0][threadIdx.y][threadIdx.x] = 0.;
		res[1][threadIdx.y][threadIdx.x] = 0.;
	} else {
		res[0][threadIdx.y][threadIdx.x] = 1.;
		res[1][threadIdx.y][threadIdx.x] = 1.;
	}
}

__device__ inline void eval_copy_matrix_to_res(struct memory * const mem,
		    	    	    	       const int cmatrix,
		    	    	    	       const int rmatrix)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int cstart = mem->c_zero + cmatrix * MATRIX_WIDTH;

	res[rmatrix][ty][tx] = C_ROW(ty)[cstart + tx];
}

__device__ void eval_mul_inplace(const struct instance * const inst,
				 struct memory         * const mem,
				 const int cmatrix,
				 const int rmatrix)
{
	const int rows = MATRIX_HEIGHT;
	const int cstart = mem->c_zero  + cmatrix * inst->dim.matrix_width;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	double tmp = 0;

	/* result rows */
	#pragma unroll
	for(int i = 0; i < rows; i++) {
		tmp += res[rmatrix][ty][i] * C_ROW(i)[cstart + tx];
	}

	__syncthreads();
	res[rmatrix][ty][tx] = tmp;
	__syncthreads();
}

__device__ const int* eval_interpret_rule(const struct instance * const inst,
				    	  struct memory		* const mem,
				    	  const int* rule,
				    	  const int  rmatrix)
{
	if(*rule == MUL_SEP)
		return rule;

	/*
	 * all multiplications are inplace,
	 * so we copy the first matrix to our result
	 */
	eval_copy_matrix_to_res(mem, *rule, rmatrix);
	rule++;

	__syncthreads();

	for(; *rule != MUL_SEP; rule++) {
		eval_mul_inplace(inst, mem, *rule, rmatrix);
	}

	return rule;
}

__shared__ struct memory res_mem;
__shared__ double shrd_rating;

__device__ double get_max_value()
{
	double my_max;
	__shared__ double max_value[MATRIX_HEIGHT];

	if(threadIdx.x == 0) {
		my_max = res[0][threadIdx.y][0];
		for(int i = 1; i < MATRIX_WIDTH; i++) {
			my_max = max(my_max, res[0][threadIdx.y][i]);
		}
		max_value[threadIdx.y] = my_max;

		__syncthreads();
		if(threadIdx.y == 0) {
			for(int k = 1; k < MATRIX_HEIGHT; k++) {
				max_value[0] = max(max_value[0], max_value[k]);
			}
		}
		__syncthreads();
	}

	return max_value[0];
}

__device__ void evo_result_rating(const struct instance * const inst,
				  struct memory         * const mem)
{
	const int rows = MATRIX_HEIGHT - 1;
	const int cols = MATRIX_WIDTH  - 1;
	double rating = 0.;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const double penalty = 1e9;

	if(ty == 0 && tx == 0) {
		switch(inst->cond_right) {
		case COND_UPPER_LEFT:
			if((res[0][0][0] - res[1][0][0]) < 1.f)
				rating += penalty;
			break;
		case COND_UPPER_RIGHT:
			if((res[0][0][cols] - res[1][0][cols]) < 1.f)
				rating += penalty;
			break;
		case COND_UPPER_LEFT_LOWER_RIGHT:
			if((res[0][0][0] - res[1][0][0]) < 1.f)
				rating += penalty;

			if((res[0][rows][cols] - res[1][rows][cols]) < 1.f)
				rating += penalty;
			break;
		default:
			rating += 2*penalty;
			break;
		}
	}

	__syncthreads();
	// keep only negative numbers
	res[0][ty][tx] = fabs(min(res[0][ty][tx] - res[1][ty][tx], 0.));

//	double max_value = get_max_value();
//	max_value = (max_value == 0 ? 1 : max_value); // div. by zero is evil...
//	res[0][ty][tx] /= max_value;
	__syncthreads();

	//only lines are processed
	if(tx != 0)
		return;

	for(int i = 1; i < MATRIX_WIDTH; i++) {
		res[0][ty][0] += res[0][ty][i];
	}

	if(ty != 0)
		return;

	for(int i = 0; i < MATRIX_HEIGHT; i++) {
		rating += res[0][i][0];
	}

	shrd_rating += rating;
}

__device__ void evo_init_mem2(const struct instance* const inst,
			      struct memory * const mem)
{
	evo_init_mem(inst, mem);
	/*
	 * each block represent one child which has a
	 * defined pos in the matrix
	 */
	mem->c_zero = inst->width_per_inst * blockIdx.y;
	mem->c_end  = inst->width_per_inst * (blockIdx.y + 1);

#ifdef DEBUG
	mem->r_zero1 = blockIdx.y * 2 * inst->dim.matrix_width;
	mem->r_end1  = mem->r_zero1 + inst->dim.matrix_width;
	mem->r_zero2 = mem->r_zero1 + inst->dim.matrix_width;
	mem->r_end2  = mem->r_zero2 + inst->dim.matrix_width;
#endif
}

__global__ void evo_calc_res(struct instance * const inst)
{
	const int* end = inst->rules + inst->rules_len - 1;
	const int* rules = inst->rules;

	char* const r_dev_ptr = (char*)inst->dev_rules.ptr;
        const size_t r_pitch = inst->dev_rules.pitch;
        const size_t r_slice_pitch = r_pitch * inst->dim.childs * inst->dim.parents;
        char* const r_slice = r_dev_ptr + blockIdx.x /* z */ * r_slice_pitch;
        uint8_t* const active_rules = (uint8_t*) (r_slice + blockIdx.y * r_pitch);

	if(threadIdx.x == 0 && threadIdx.y == 0) {
		evo_init_mem2(inst, &res_mem);
		shrd_rating = 0.;
	}

	__syncthreads();
	uint8_t cur_rule = 0;

	do {
		/* ignore matched rules */	
		if(inst->match == MATCH_ANY && !active_rules[cur_rule]) {
			rules++;
			while(*rules != MUL_SEP) {
				rules++;
			}
			rules++;
			while(*rules != MUL_SEP) {
				rules++;
			}
			cur_rule++;
			__syncthreads();
			continue;
		}

		eval_set_res_matrix_to_identity();

		rules++;
		rules = eval_interpret_rule(inst , &res_mem, rules, 0);

		rules++;
		rules = eval_interpret_rule(inst , &res_mem, rules, 1);

		const double old_rating = shrd_rating;
		evo_result_rating(inst, &res_mem);
		__syncthreads();

		if(inst->match == MATCH_ANY && old_rating == shrd_rating) {
			active_rules[cur_rule] = 0;
		}

		#ifdef DEBUG
		if(shrd_rating == 0.) {
			struct memory *mem = &res_mem;
			for(int i = 0; i < inst->num_matrices; i++) {
				R_ROW(ty)[tx + i * MATRIX_WIDTH] =
						C_ROW(ty)[res_mem.c_zero + i * MATRIX_WIDTH + tx];
			}

			inst->res_child_block = blockIdx.x;
			inst->res_child_idx   = blockIdx.y;
		}
		#endif

		cur_rule++;
		__syncthreads();
	} while(rules != end);

	__syncthreads();

	if(threadIdx.x == 0 && threadIdx.y == 0) {
		res_mem.c_rat[2 * blockIdx.y]     = shrd_rating;
		res_mem.c_rat[2 * blockIdx.y + 1] = blockIdx.y;
	}
}
