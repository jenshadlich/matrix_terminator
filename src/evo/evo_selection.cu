#include "hip/hip_runtime.h"
/*
 * evo_selection.cu
 *
 *  Created on: Jun 24, 2011
 *      Author: tkalbitz
 */

__device__ void evo_parent_selection_best(struct instance * const inst,
					  struct memory   * const mem)
{
	if(ty != 0)
		return;

	__shared__ struct double2 res[PARENTS * CHILDS];
	double2* const arr   = (double2*)mem->c_rat;

	for(int i = tx; i < PARENTS * CHILDS; i += inst->dim.matrix_width) {
		res[i]   = arr[i];
	}

	__syncthreads();

	if(tx == 0) {
		double2 key;

		/* insertion sort */
		for(int i = 1; i < PARENTS * CHILDS; i++) {
			key = res[i];

			int j = i - 1;
			while(j >=0 && res[j].x > key.x) {
				res[j + 1] = res[j];
				j = j - 1;
			}
			res[j + 1] = key;
		}
	}

	__syncthreads();

	for(int i = tx; i < PARENTS * CHILDS; i += inst->dim.matrix_width) {
		arr[i] = res[i];
	}
}

__device__ void evo_parent_selection_turnier(struct instance * const inst,
		                             struct memory   * const mem,
					     hiprandState* rnd_state,
					     const uint8_t q)
{
	__shared__ struct double2 src[PARENTS * CHILDS];
	__shared__ struct double2 dest[PARENTS];
	double2* const arr = (double2*)mem->c_rat;

	if(threadIdx.y == 0 && threadIdx.x < PARENTS) {
		for(int i = tx; i < PARENTS * CHILDS; i += inst->dim.matrix_width) {
			src[i]   = arr[i];
		}
	}

	__syncthreads();

	if(threadIdx.y == 0 && threadIdx.x < PARENTS) {
		for(int pos = tx; pos < PARENTS; pos += inst->dim.matrix_width) {
			uint32_t idx = hiprand(rnd_state) % (PARENTS * CHILDS);
			for(uint8_t t = 0; t < q; t++) {
				uint32_t opponent = hiprand(rnd_state) % (PARENTS * CHILDS);

				if(src[opponent].x < src[idx].x)
					idx = opponent;
			}

			dest[pos] = src[idx];
		}
	}

	__syncthreads();

	if(threadIdx.x == 0) {
		double2 key;

		/* insertion sort */
		for(int i = 1; i < PARENTS; i++) {
			key = dest[i];

			int j = i - 1;
			while(j >=0 && dest[j].x > key.x) {
				dest[j + 1] = dest[j];
				j = j - 1;
			}
			dest[j + 1] = key;
		}

	}

	__syncthreads();

	if(threadIdx.y == 0 && threadIdx.x < PARENTS) {
		for(int i = tx; i < PARENTS; i += inst->dim.matrix_width) {
			arr[i] = dest[i];
		}
	}
}

__device__ void evo_parent_selection_convergence_prevention(
					     struct instance * const inst,
		                             struct memory   * const mem,
					     hiprandState* rnd_state,
					     const float cp)
{
	if(ty != 0)
		return;

	__shared__ struct double2 res[PARENTS * CHILDS];

	double2* const arr   = (double2*)mem->c_rat;

	for(int i = tx; i < PARENTS * CHILDS; i += inst->dim.matrix_width) {
		res[i]   = arr[i];
	}

	__syncthreads();

	double2 key;

	for(int k = 64; k < NEXT_2POW; k *= 2) {
		for(int p = k * tx; p < PARENTS * CHILDS; p += k * inst->dim.matrix_width) {
			const int end =  min(k * (tx + 1), PARENTS * CHILDS);

			/* insertion sort */
			for(int i = p + 1; i < end; i++) {
				key = res[i];

				int j = i - 1;
				while(j >=0 && res[j].x > key.x) {
					res[j + 1] = res[j];
					j = j - 1;
				}
				res[j + 1] = key;
			}
		}
		__syncthreads();
	}

	if(tx == 0) {
//		double2 key;
//
//		/* insertion sort */
//		for(int i = 1; i < PARENTS * CHILDS; i++) {
//			key = res[i];
//
//			int j = i - 1;
//			while(j >=0 && res[j].x > key.x) {
//				res[j + 1] = res[j];
//				j = j - 1;
//			}
//			res[j + 1] = key;
//		}
//
		int last = 0;
		for(int i = 1; i < PARENTS * CHILDS; i++) {
			if(res[last].x == res[i].x) {
				if(hiprand_normal(rnd_state) < cp) {
					res[i].x = FLT_MAX;
				}
			} else {
				last = i;
			}
		}

		/* insertion sort */
		for(int i = 1; i < PARENTS * CHILDS; i++) {
			key = res[i];

			/*
			 * we need only parents count and know that
			 * the array was already sorted
			 */
			if(i > PARENTS && res[PARENTS - 1].x < FLT_MAX)
				break;

			int j = i - 1;
			while(j >=0 && res[j].x > key.x) {
				res[j + 1] = res[j];
				j = j - 1;
			}
			res[j + 1] = key;
		}
	}
	__syncthreads();

	for(int i = tx; i < PARENTS * CHILDS; i += inst->dim.matrix_width) {
		arr[i] = res[i];
	}
}
