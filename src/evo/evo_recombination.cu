#include "hip/hip_runtime.h"
/*
 * Select two parents for recombination.
 * Selection is currently complete uniform.
 */
inline __device__ void evo_recomb_selection(const struct instance* const inst,
					    hiprandState* const rnd_state,
					    int* const sel)
{
	sel[0] = hiprand(rnd_state) % inst->dim.parents;
	sel[1] = hiprand(rnd_state) % inst->dim.parents;
}

/* A uniform crossover recombination. */
__device__ void evo_recombination(struct instance * const inst,
				  struct memory   * const mem,
				  hiprandState     * const rnd_state,
				  const int       * const sel)
{
	const int rows = inst->dim.matrix_height;
	const int cols = inst->width_per_inst;

	const int p1   = sel[0] * inst->width_per_inst;
	const int p2   = sel[1] * inst->width_per_inst;
	const int cIdx = mem->c_zero;

	RR(tx) = ((PRR(sel[0]) + PRR(sel[1])) / 2.) + (hiprand_normal(rnd_state) / 100);
	RR(tx) = min(max(RR(tx), 0.5), 1.);
//	RR(tx) = PRR(sel[0]);

	MR(tx) = (1 - RR(tx)) * PMR(sel[0]) + RR(tx) * PMR(sel[1]);
	SP(tx) = (1 - RR(tx)) * PSP(sel[0]) + RR(tx) * PSP(sel[1]);

	const double mr = RR(tx);

	for(int r = 0; r < rows; r++) {
		double* const c_row = C_ROW(r);
		double* const p_row = P_ROW(r);

		for(int c = 0; c < cols; c++) {
			if(hiprand_uniform(rnd_state) > mr) {
				c_row[cIdx + c] = p_row[p1 + c];
			} else {
				c_row[cIdx + c] = p_row[p2 + c];
			}
		}
	}
}
