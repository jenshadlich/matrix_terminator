#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "instance.h"
#include "evo_setup.h"

int get_evo_threads(const struct instance * const inst)
{
	return inst->dim.parents * inst->dim.childs;
}

/*
 * Allocate memory for the parent matrices. the memory is layouted for faster
 * access. The block count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_parent_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents    * /* there are n parents per block */
		    inst->width_per_inst *
		    sizeof(double);

	inst->dev_parent_ext = make_hipExtent(width,
					       inst->dim.matrix_height,
					       inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_parent_ext));

	inst->dev_parent = pitched_ptr;
}

/*
 * Allocate memory for the child matrices. the memory is layouted for faster
 * access. The bloc count is the depth of the allocated memory. All threads of
 * one block had to operate on a part of the width.
 */
void alloc_child_matrix(struct instance *inst)
{
	assert(inst->num_matrices != 0);

	int width = inst->dim.parents * inst->dim.childs * /* each parent should have n childs */
		    inst->width_per_inst * sizeof(double);

	inst->dev_child_ext = make_hipExtent(width,
					      inst->dim.matrix_height,
					      inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_child_ext));
	inst->dev_child = pitched_ptr;
}

/**
 * Allocate the matrix for each thread which is
 * needed for the multiplication and evaluation.
 */
void alloc_result_matrix(struct instance *inst)
{
	const int width = inst->dim.childs * inst->dim.parents *
			    inst->dim.matrix_width * sizeof(double);

	inst->dev_res_ext = make_hipExtent(width,
					    inst->dim.matrix_height,
					    inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_res_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 1, inst->dev_res_ext));
	inst->dev_res = pitched_ptr;
}

void alloc_sparam(struct instance *inst)
{
	inst->dev_sparam_ext = make_hipExtent(inst->dim.childs *
					       inst->dim.parents * 3 *
					       sizeof(double),
					       1,
					       inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_sparam_ext));
	inst->dev_sparam = pitched_ptr;

	inst->dev_psparam_ext = make_hipExtent(inst->dim.parents * 3 *
						sizeof(double),
					        1,
					        inst->dim.blocks);

	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_psparam_ext));
	inst->dev_psparam = pitched_ptr;
}

void alloc_rating(struct instance *inst)
{
	inst->dev_crat_ext = make_hipExtent(2 * get_evo_threads(inst) * sizeof(double),
	 			    	     1,
	 			    	     inst->dim.blocks);

	inst->dev_prat_ext = make_hipExtent(inst->dim.parents * sizeof(double),
					     1,
					     inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_crat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 33, inst->dev_crat_ext));
	inst->dev_crat = pitched_ptr;

	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_prat_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 33, inst->dev_prat_ext));
	inst->dev_prat = pitched_ptr;
}

void alloc_debug_matrix(struct instance *inst)
{
#ifdef DEBUG
	/* memory for every rule side of every rule */
	const int width = inst->dim.childs * inst->dim.parents *
			    inst->dim.matrix_width * inst->rules_count *
			    2 * sizeof(double);
#else
	const int width = 1;
#endif

	inst->dev_debug_ext = make_hipExtent(width,
					    inst->dim.matrix_height,
					    inst->dim.blocks);

	hipPitchedPtr pitched_ptr;
	CUDA_CALL(hipMalloc3D(&pitched_ptr, inst->dev_debug_ext));
	CUDA_CALL(hipMemset3D(pitched_ptr, 1, inst->dev_debug_ext));
	inst->dev_debug = pitched_ptr;
}

void init_rnd_generator(struct instance *inst, int seed)
{
	hiprandState *rnd_states;
	const int count = max(get_evo_threads(inst), inst->dim.matrix_height);

	CUDA_CALL(hipMalloc((void **)&rnd_states,
			     count * BLOCKS * sizeof(hiprandState)));
	setup_rnd_kernel<<<BLOCKS, count>>>(rnd_states, seed);
	CUDA_CALL(hipGetLastError());
	hipDeviceSynchronize();
	inst->rnd_states = rnd_states;
}

void set_num_matrices(struct instance* inst)
{
	int m = INT_MIN;
	for(int i = 0; i < inst->rules_len; i++)
		m = max(m, inst->rules[i]);

	inst->num_matrices = m + 1; /* matrices are zero based */
//	printf("num_matrices set to %d\n", inst->num_matrices);
}

void inst_init(struct instance* const inst, int matrix_width)
{
	inst->dim.blocks  = BLOCKS;
	inst->dim.childs  = CHILDS;
	inst->dim.parents = PARENTS;
	inst->dim.matrix_width  = matrix_width;
	inst->dim.matrix_height = matrix_width;

	inst->res_block = 0;
	inst->res_parent = 0;
	inst->res_child_block = 0;
	inst->res_child_idx = 0;

	set_num_matrices(inst);

	inst->width_per_inst = inst->num_matrices *    /* there are n matrices needed for the rules */
			       inst->dim.matrix_width; /* each one has a fixed width */

	alloc_parent_matrix(inst);
	alloc_child_matrix(inst);
	alloc_result_matrix(inst);
	alloc_rating(inst);
	alloc_sparam(inst);
	alloc_debug_matrix(inst);
	init_rnd_generator(inst, time(0));
}

void inst_cleanup(struct instance * const inst,
		  struct instance * const dev_inst)
{
	if(dev_inst != NULL)
		hipFree(dev_inst);

	hipFree(inst->rnd_states);
	hipFree(inst->dev_child.ptr);
	hipFree(inst->dev_parent.ptr);
	hipFree(inst->dev_res.ptr);
	hipFree(inst->dev_crat.ptr);
	hipFree(inst->dev_prat.ptr);
	hipFree(inst->dev_sparam.ptr);
	hipFree(inst->dev_debug.ptr);
}

struct instance* inst_create_dev_inst(struct instance *inst, int** dev_rules)
{
	struct instance *dev_inst;
	int *rules = inst->rules;
	int *tmp_dev_rules;
	CUDA_CALL(hipMalloc(&tmp_dev_rules, inst->rules_len * sizeof(int)));
	CUDA_CALL(hipMemcpy(tmp_dev_rules,  rules, inst->rules_len * sizeof(int),
					hipMemcpyHostToDevice));
	inst->rules = tmp_dev_rules;
	CUDA_CALL(hipMalloc(&dev_inst, sizeof(*dev_inst)));
	CUDA_CALL(hipMemcpy(dev_inst, inst, sizeof(*dev_inst),
					hipMemcpyHostToDevice));
	inst->rules = rules;

	if(dev_rules != NULL)
		*dev_rules = tmp_dev_rules;

	return dev_inst;
}

void inst_copy_dev_to_host(struct instance * const dev,
			   struct instance * const host)
{
	int *rules = host->rules;
	CUDA_CALL(hipMemcpy(host, dev, sizeof(*dev), hipMemcpyDeviceToHost));
	host->rules = rules;
}

