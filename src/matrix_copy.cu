#include "matrix_copy.h"

void copy_parents_dev_to_host(struct instance* inst, void* parent_cpy)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr = inst->dev_parent;
	p.dstPtr = make_hipPitchedPtr(
			parent_cpy,
			inst->dev_parent_ext.width,
			inst->dev_parent_ext.width / sizeof(float),
			inst->dim.matrix_height);

	p.extent = inst->dev_parent_ext;
	p.kind = hipMemcpyDeviceToHost;
	CUDA_CALL(hipMemcpy3D(&p));
}

void copy_parent_rating_dev_to_host(struct instance* inst, void* parent_rat_cpy)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr = inst->dev_prat;
	p.dstPtr = make_hipPitchedPtr(
			parent_rat_cpy,
			inst->dim.parents * sizeof(float),
			inst->dim.parents,
			1);

	p.extent = inst->dev_prat_ext;
	p.kind = hipMemcpyDeviceToHost;
	CUDA_CALL(hipMemcpy3D(&p));
}

void copy_results_dev_to_host(struct instance* inst, void* result_cpy)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr = inst->dev_res;
	p.dstPtr = make_hipPitchedPtr(
			result_cpy,
			inst->dev_res_ext.width,
			inst->dev_res_ext.width / sizeof(float),
			inst->dim.matrix_height);

	p.extent = inst->dev_res_ext;
	p.kind = hipMemcpyDeviceToHost;
	CUDA_CALL(hipMemcpy3D(&p));
}
