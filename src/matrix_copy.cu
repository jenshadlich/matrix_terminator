#include "matrix_copy.h"

void copy_parents_dev_to_host(struct instance* inst, void* parent_cpy)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr = inst->dev_parent;
	p.dstPtr = make_hipPitchedPtr(
			(void*) parent_cpy,
			inst->dim.threads * inst->width_per_inst * sizeof(float),
			inst->dim.threads * inst->width_per_inst,
			inst->dim.matrix_height);

	p.extent = inst->dev_parent_ext;
	p.kind = hipMemcpyDeviceToHost;
	CUDA_CALL(hipMemcpy3D(&p));
}
