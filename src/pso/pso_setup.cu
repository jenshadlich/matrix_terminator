#include "hip/hip_runtime.h"
/*
 * pso_setup.cu
 *
 *  Created on: Sep 28, 2011
 *      Author: tkalbitz
 */

#include <float.h>

#include "pso_config.h"
#include "pso_setup.h"
#include "pso_memory.h"

__device__ static double evo_mut_new_value(struct pso_instance * const inst,
					   hiprandState         * const rnd_state)
{
	/* we want to begin with small numbers */
	const int tmp = (inst->parent_max > 10) ? 10 : (int)inst->parent_max;
	const int rnd_val = (hiprand(rnd_state) % (tmp - 1)) + 1;
	int factor = (int)(rnd_val / inst->delta);
	if((factor * inst->delta) < 1.0)
		factor*=2;

	const double val = factor * inst->delta;
	if(val < 1.0)
		return 1.0;

	return val;
}

__global__ void setup_rnd_kernel(hiprandState* const rnd_states,
				 const int seed)
{
	const int id = get_thread_id();

	/*
         * Each thread get the same seed,
         * a different sequence number and no offset.
         */
	hiprand_init(seed + id, id, 0, &rnd_states[id]);
}

/*
 * Initialize the child memory with random values.
 */
__global__ void
setup_particle_kernel(struct pso_instance * const inst, bool half)
{
	const int id = get_thread_id();
	hiprandState rnd = inst->rnd_states[id];

	const int max1 = (int)inst->parent_max;
	const int end = inst->total;
	int x;

	for(x = tx; x < end; x += blockDim.x) {
		if(hiprand_uniform(&rnd) < MATRIX_TAKEN_POS) {
			inst->particle[x] = hiprand(&rnd) % max1 ;
		} else {
			inst->particle[x] = 0;
		}
	}

	__syncthreads();

	const int matrices = inst->num_matrices *
			     inst->dim.particles *
			     inst->dim.blocks;

	if(inst->cond_left == COND_UPPER_LEFT) {
		for(x = tx; x < matrices; x += blockDim.x) {
			const int matrix = x * inst->width_per_matrix;
                        inst->particle[matrix] = evo_mut_new_value(inst, &rnd);
		}
	} else if(inst->cond_left == COND_UPPER_RIGHT) {
		for(x = tx; x < matrices; x += blockDim.x) {
			const int matrix = x * inst->width_per_matrix +
					   inst->dim.matrix_width - 1;
                        inst->particle[matrix] = evo_mut_new_value(inst, &rnd);
		}
	} else if(inst->cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
		for(x = tx; x < matrices; x += blockDim.x) {
			const int matrix1 = x * inst->width_per_matrix;
			const int matrix2 = (x + 1) * inst->width_per_matrix - 1;
                        inst->particle[matrix1] = evo_mut_new_value(inst, &rnd);
                        inst->particle[matrix2] = evo_mut_new_value(inst, &rnd);
		}
	}
	inst->rnd_states[id] = rnd;
	__syncthreads();
}

__global__ void setup_rating(struct pso_instance * const inst)
{
	int i = 0;
	int len = (inst->width_per_line / 2 + 1) * inst->dim.particles *
		   inst->dim.blocks;

	for(i = tx; i < len; i += blockDim.x) {
		inst->prat[i] = FLT_MAX;
		inst->lbrat[i] = FLT_MAX;
	}

	const int end = inst->dim.blocks;
	if(tx < end) {
		inst->gb_best[tx] = FLT_MAX;
		inst->gb_old[tx]  = FLT_MAX;
	}

	//TODO
	if(tx < BLOCKS) {
		inst->s[tx] = 2;
	}
}

__global__ void setup_col_permut(int* const col_permut,
		                 const int total,
		                 const int width_per_line)
{
	int i;

	for(i = tx; i < total; i += blockDim.x) {
		col_permut[i] = (i % width_per_line);
	}
}
