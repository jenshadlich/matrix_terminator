#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>

#include <hiprand/hiprand_kernel.h>

#include "pso.h"
#include "pso_config.h"
#include "pso_memory.h"

#if __CUDA_ARCH__ >= 200
	#define DIV(x,y) __ddiv_rn((x),(y))
#else
	#define DIV(x,y) ((x)/(y))
#endif

#define PRED_STEP(s) if(tx < (s)) { \
			if(shm_rat[tx] > shm_rat[tx + (s)]) { \
				shm_rat[tx] = shm_rat[tx + (s)]; \
				shm_pos[tx] = shm_pos[tx + (s)]; \
			}}

/**
 * can only be launched with PARTICLE_COUNT threads
 */
__global__ void pso_evaluation_lbest(const struct pso_instance inst,
				     const int s,
				     const int cur)
{
	__shared__ double shm_rat[PARTICLE_COUNT];
	__shared__ int    shm_pos[PARTICLE_COUNT];

	const int block_pos = BLOCK_POS;
	const int s_count = inst.width_per_line / s;
	const int c = cur / PARTICLE_COUNT;
	const int * const col_permut = inst.col_permut + inst.width_per_line * bx;

	const double * const prat = inst.prat  + s_count * bx * PARTICLE_COUNT;
	double * const lbrat      = inst.lbrat + s_count * bx * PARTICLE_COUNT;

	double* const particle = inst.particle;
	double* const particle_lbest = inst.particle_lbest + bx * s_count;
	double* const particle_gbest = inst.particle_gbest + bx * inst.width_per_line;

	//copy rating to shm
	shm_rat[tx] = lbrat[cur + tx];
	shm_pos[tx] = tx;

	//new particle is better than his memory
	if(prat[cur + tx] < shm_rat[tx]) {
		for(int j = 0; j < s; j++) {
			const int idx = ELEM_BIDX(block_pos, tx, col_permut[c * s + j]);
			particle_lbest[idx + j] = particle[idx + j];
		}

		shm_rat[tx] = lbrat[cur + tx] = prat[cur + tx];
	}

	__syncthreads();

	//reduction step
//	if (PARTICLE_COUNT >= 256) { PRED_STEP(128); __syncthreads(); }
//	if (PARTICLE_COUNT >= 128) { PRED_STEP(64);  __syncthreads(); }
//	if (PARTICLE_COUNT >=  64) { PRED_STEP(32);  __syncthreads(); }
	if (PARTICLE_COUNT >=  32) { PRED_STEP(16);  __syncthreads(); }
	if (PARTICLE_COUNT >=  16) { PRED_STEP( 8);  __syncthreads(); }
	if (PARTICLE_COUNT >=   8) { PRED_STEP( 4);  __syncthreads(); }
	if (PARTICLE_COUNT >=   4) { PRED_STEP( 2);  __syncthreads(); }
	if (PARTICLE_COUNT >=   2) { PRED_STEP( 1);  __syncthreads(); }

	__syncthreads();

	//copy step
	if(shm_rat[0] < inst.gbrat[bx]) {
		for(int j = tx; j < s; j += blockDim.x) {
			const int col = col_permut[c * s + j];
			const int idx = ELEM_BIDX(block_pos, shm_pos[0], col);
			particle_gbest[col] = particle_lbest[idx + j];
		}

		if(tx == 0)
			inst.gbrat[bx] = shm_rat[0];
	}
}

__global__ void pso_neighbor_best(const struct pso_instance inst)
{
	const int s = inst.s[bx];
	const int s_count = inst.width_per_line / s;

	double * const lbrat  = inst.lbrat     + s_count * bx * PARTICLE_COUNT;
	int    * const lb_idx = inst.lbest_idx + s_count * bx * PARTICLE_COUNT;

	int n_particle = tx + 1;
	int p_particle = tx - 1;

	if(n_particle == PARTICLE_COUNT) {
		n_particle = 0;
	}

	if(p_particle == -1) {
		p_particle = PARTICLE_COUNT - 1;
	}

	for(int i = 0; i < s_count; i++) {
		const double lb_rat_p = lbrat[i * PARTICLE_COUNT + p_particle];
		const double lb_rat_c = lbrat[i * PARTICLE_COUNT + tx];
		const double lb_rat_n = lbrat[i * PARTICLE_COUNT + n_particle];
		const double res = min(min(lb_rat_p, lb_rat_c), lb_rat_n);

		int particle;

		if(res == lb_rat_p) {
			particle = p_particle;
		} else if(res == lb_rat_c) {
			particle = tx;
		} else if(res == lb_rat_n) {
			particle = n_particle;
		}

		lb_idx[i * PARTICLE_COUNT + tx] = particle;
	}
}

__device__ float curand_cauchy(hiprandState* rnd)
{
	float v = 0.0f;

	do {
		v = hiprand_normal(rnd);
	} while(v == 0);

	return hiprand_normal(rnd) / v;
//	return tan(M_PI * hiprand_uniform(rnd));
}

__device__ static double pso_mut_new_value(const struct pso_instance & inst,
					   hiprandState         * const rnd_state)
{
	/* we want to begin with small numbers */
	const int tmp = (inst.parent_max > 10) ? 10 : (int)inst.parent_max;

	const int rnd_val = (hiprand(rnd_state) % (tmp - 1)) + 1;
	int factor = (int)(rnd_val / inst.delta);
	if((factor * inst.delta) < 1.0)
		factor++;

	if(factor * inst.delta < 1.0)
		return 1;

	return factor * inst.delta;
}

__device__ void pso_ensure_constraints(const struct pso_instance & inst,
				       hiprandState         * const rnd,
				       double              * const elems)
{
	const int matrices = inst.num_matrices;
	int x;

	if(tx < PARTICLE_COUNT) {
		if(inst.cond_left == COND_UPPER_LEFT) {
			for(x = 0; x < matrices; x++) {
				const int matrix = x * inst.width_per_matrix * PARTICLE_COUNT + tx;

				if(elems[matrix] < 1.0)
					elems[matrix] = pso_mut_new_value(inst, rnd);
			}
		} else if(inst.cond_left == COND_UPPER_RIGHT) {
			for(x = 0; x < matrices; x++) {
				const int matrix = x * inst.width_per_matrix * PARTICLE_COUNT +
						   inst.dim.matrix_width * PARTICLE_COUNT - 1 - tx;
				if(elems[matrix] < 1.0)
					elems[matrix] = pso_mut_new_value(inst, rnd);
			}
		} else if(inst.cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
			for(x = 0; x < matrices; x++) {
				const int matrix1 = x * inst.width_per_matrix * PARTICLE_COUNT + tx;
				const int matrix2 = (x + 1) * inst.width_per_matrix * PARTICLE_COUNT - 1 - tx;

				if(elems[matrix1] < 1.0)
					elems[matrix1] = pso_mut_new_value(inst, rnd);

				if(elems[matrix2] < 1.0)
					elems[matrix2] = pso_mut_new_value(inst, rnd);
			}
		} else {
			/*
			 * This should be recognized ;) It's only a 1.3 card
			 *  so there is no printf :/
			 */
			for(int i = 0; i < inst.width_per_matrix * PARTICLE_COUNT; i++) {
				elems[i] = 1337;
			}
		}
	}
}

__global__ void pso_swarm_step_ccpso2(const struct pso_instance inst)
{
	const int s = inst.s[bx];

	const double delta = inst.delta;
	const int id = get_thread_id();
	hiprandState rnd_state = inst.rnd_states[id];

	double* const elems = inst.particle;
	double* const particle_lbest = inst.particle_lbest;


	const int* const col_perm = inst.col_permut + bx * inst.width_per_line;
	const int col_start = tx / PARTICLE_COUNT;
	const int col_add   = blockDim.x / PARTICLE_COUNT;
	const int end	    = inst.width_per_line;
	const int particle  = tx - col_start * PARTICLE_COUNT;
	const int block_pos = BLOCK_POS;

	const int* const lb_idx = inst.lbest_idx + (end / s) * bx * PARTICLE_COUNT;

	for(int i = col_start; i < end; i += col_add) {
		const int idx   = ELEM_BIDX(block_pos, particle, col_perm[i]);
		const int cur_s = i / s;
		const double lb  = particle_lbest[idx];
		const double lbn = particle_lbest[ELEM_BIDX(block_pos, lb_idx[cur_s], col_perm[i])];

		double xi = elems[idx];

		if(hiprand_uniform(&rnd_state) <= 0.5)
			xi = lb  + curand_cauchy(&rnd_state) * abs((lb - lbn));
		else
			xi = lbn + hiprand_normal(&rnd_state) * abs((lb - lbn));

		xi = __dmul_rn(__double2uint_rn(DIV(xi, delta)), delta);
		xi = min(inst.parent_max, max(0., xi));

		elems[idx] = xi;
	}

	__syncthreads();

	if(tx < PARTICLE_COUNT) {
		const int idx   = ELEM_BIDX(block_pos, 0, 0);
		pso_ensure_constraints(inst, &rnd_state, elems + idx);
	}

	inst.rnd_states[id] = rnd_state;
}
