#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, 2012 Tobias Kalbitz <tobias.kalbitz@googlemail.com>
 *
 * All rights reserved. This program and the accompanying materials
 * are made available under the terms of the GNU Public License v2.0
 * which accompanies this distribution, and is available at
 * http://www.gnu.org/licenses/old-licenses/gpl-2.0.html
 */

#include <float.h>

#include <hiprand/hiprand_kernel.h>

#include "pso.h"
#include "pso_config.h"
#include "pso_memory.h"

#if __CUDA_ARCH__ >= 200
	#define DIV(x,y) __ddiv_rn((x),(y))
#else
	#define DIV(x,y) ((x)/(y))
#endif

__global__ void pso_evaluation_lbest(struct pso_instance* inst)
{
	struct memory m;
	struct memory* mem = &m;
	pso_init_mem(inst, mem);

	if(mem->lb_rat[blockIdx.y] > mem->p_rat[blockIdx.y]) {
		for(int i = 0; i < inst->num_matrices; i++) {
			int delta = mem->p_zero + i * inst->dim.matrix_width;
			LB_ROW(ty)[delta + tx] = P_ROW(ty)[delta + tx];
		}

		if(tx == 0 && ty == 0)
			mem->lb_rat[blockIdx.y] = mem->p_rat[blockIdx.y];
	}
}

__global__ void pso_evaluation_gbest(struct pso_instance* inst)
{
	struct memory m;
	struct memory* mem = &m;
	pso_init_mem(inst, mem);

	__shared__ int pidx;

	if(tx == 0 && ty == 0) {
		pidx = -1;

		double rat = inst->gb_rat[blockIdx.x];
		for(int i = 0; i < PARTICLE_COUNT; i++) {
			if(rat > mem->lb_rat[i]) {
				rat = mem->lb_rat[i];
				pidx = i;
			}
		}
	}

	__syncthreads();

	if(pidx == -1)
		return;

	int src = pidx * inst->width_per_inst;
	for(int i = 0; i < inst->num_matrices; i++) {
		int delta = i * inst->dim.matrix_width;
		GB_ROW(ty)[delta + tx] = LB_ROW(ty)[src + delta + tx];
	}

	if(tx == 0 && ty == 0)
		inst->gb_rat[blockIdx.x] = mem->lb_rat[pidx];
}


__device__ static double pso_mut_new_value(struct pso_instance * const inst,
					   hiprandState         * const rnd_state)
{
	/* we want to begin with small numbers */
	const int tmp = (inst->parent_max > 10) ? 10 : (int)inst->parent_max;

	const int rnd_val = (hiprand(rnd_state) % (tmp - 1)) + 1;
	int factor = (int)(rnd_val / inst->delta);
	if((factor * inst->delta) < 1.0)
		factor++;

	if(factor * inst->delta < 1.0)
		return 1;

	return factor * inst->delta;
}

__device__ void pso_ensure_constraints(struct pso_instance * const inst,
				       struct memory       * const mem,
				       hiprandState         * const rnd_state)
{
	double* const row   = P_ROW(0);
	double* const lrow  = P_ROW(inst->dim.matrix_height-1);

	const int end = mem->p_end;

	for(int start = mem->p_zero; start < end; start += inst->dim.matrix_width) {
		const int lidx = start + inst->dim.matrix_width - 1;

		if(inst->cond_left == COND_UPPER_LEFT) {
			if(row[start] < 1.0)
			row[start] = pso_mut_new_value(inst, rnd_state);
		} else if(inst->cond_left == COND_UPPER_RIGHT) {
			if(row[lidx] < 1.0)
				row[lidx] = pso_mut_new_value(inst, rnd_state);
		} else if(inst->cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
			if(row[start] < 1.0)
				row[start] = pso_mut_new_value(inst, rnd_state);

			if(lrow[lidx] < 1.0)
				lrow[lidx] = pso_mut_new_value(inst, rnd_state);
		} else {
			/*
			 * This should be recognized ;) It's only a 1.3 card
			 *  so there is no printf :/
			 */
			for(int i = 0; i < inst->dim.matrix_width; i++) {
				row[start + i] = 1337;
				lrow[start + i] = 1337;
			}
		}
	}
}


__device__ void pso_neighbor_best(struct pso_instance* const inst,
		                  struct memory*       const mem)
{
	int n_block    = blockIdx.x;
	int n_particle = blockIdx.y + 1;
	int p_block    = blockIdx.x;
	int p_particle = blockIdx.y - 1;

	if(n_particle == PARTICLE_COUNT) {
		n_particle = 0;
		n_block++;

		if(n_block == BLOCKS)
			n_block = 0;
	}

	if(p_particle == -1) {
		p_particle = PARTICLE_COUNT - 1;
		p_block--;

		if(p_block == -1)
			p_block = BLOCKS - 1;
	}

	const char* const lbrat_ptr = (char*)inst->dev_lbrat.ptr;
	const double lb_rat_p = ((double*)(lbrat_ptr + p_block    * inst->dev_lbrat.pitch))[p_particle];
	const double lb_rat_c = ((double*)(lbrat_ptr + blockIdx.x * inst->dev_lbrat.pitch))[blockIdx.y];
	const double lb_rat_n = ((double*)(lbrat_ptr + n_block    * inst->dev_lbrat.pitch))[n_particle];

	const double res = min(min(lb_rat_p, lb_rat_c), lb_rat_n);

	int block;
	int particle;

	if(res == lb_rat_p) {
		block = p_block;
		particle = p_particle;
	} else if(res == lb_rat_c) {
		block = blockIdx.x;
		particle = blockIdx.y;
	} else if(res == lb_rat_n) {
		block = n_block;
		particle = n_particle;
	}

	char* const  lbest_dev_ptr = (char*)inst->dev_particle_lbest.ptr;
	const size_t lbest_pitch = inst->dev_particle_lbest.pitch;
	const size_t lbest_slice_pitch = lbest_pitch * inst->dim.matrix_height;
	char* const  lbest_slice = lbest_dev_ptr + block /* z */ * lbest_slice_pitch;
	mem->lbn_pitch = lbest_pitch;
	mem->lbn_slice = lbest_slice;
	mem->lbn_zero = inst->width_per_inst * particle;
}

__global__ void pso_swarm_step(struct pso_instance* inst)
{
	__shared__ struct memory m;
	__shared__ double w;
	__shared__ double c1;
	__shared__ double c2;

	struct memory* mem = &m;

	int id = get_thread_id();
	hiprandState rnd_state = inst->rnd_states[id];

	if(tx == 0 && ty == 0) {
		pso_init_mem(inst, mem);
		pso_neighbor_best(inst, mem);
		w = W(blockIdx.y);
		c1 = C1(blockIdx.y);
		c2 = C2(blockIdx.y);
	}
	__syncthreads();

	const double delta = inst->delta;

	for(int i = 0; i < inst->num_matrices; i++) {
		const int e_idx = i * inst->dim.matrix_width + tx;
		const int p_idx = mem->p_zero + e_idx;
		const int n_idx = mem->lbn_zero + e_idx;

		double xi = P_ROW(ty)[p_idx];

		const double cog_part = hiprand_normal(&rnd_state) * c1 * (LB_ROW(ty) [p_idx] - xi);
		const double soc_part = hiprand_normal(&rnd_state) * c2 * (LBN_ROW(ty)[n_idx] - xi);

//		if(blockIdx.x < (BLOCKS / 2))
//			soc_part = hiprand_normal(&rnd_state) * c2 * (GB_ROW(ty)[e_idx] - xi);

		double tmp = max(inst->parent_max / 4, inst->delta);
		V_ROW(ty)[p_idx] = w * (V_ROW(ty)[p_idx] + cog_part + soc_part);
		V_ROW(ty)[p_idx] = min(max(V_ROW(ty)[p_idx], -tmp), tmp);

		xi = __dadd_rn(xi, V_ROW(ty)[p_idx]);
		/* we want x * delta, where x is an int */
		xi = __dmul_rn(((unsigned long)DIV(xi, delta)), delta);
		xi = min(inst->parent_max, max(0., xi));
		P_ROW(ty)[p_idx] = xi;
	}

	__syncthreads();

	if(tx == 0 && ty == 0) {
		pso_ensure_constraints(inst, mem, &rnd_state);
	}

	inst->rnd_states[id] = rnd_state;
}

__device__ float curand_cauchy(hiprandState* rnd)
{
	float v = 0.0f;

	do {
		v = hiprand_normal(rnd);
	} while(v == 0);

	return hiprand_normal(rnd) / v;
//	return tan(M_PI * hiprand_uniform(rnd));
}

__global__ void pso_swarm_step_ccpso(struct pso_instance* inst)
{
	__shared__ struct memory m;
	struct memory* const mem = &m;
	const double delta = inst->delta;
	const int id = get_thread_id();
	hiprandState rnd_state = inst->rnd_states[id];

	if(tx == 0 && ty == 0) {
		pso_init_mem(inst, mem);
		pso_neighbor_best(inst, mem);
	}
	__syncthreads();

	for(int i = 0; i < inst->num_matrices; i++) {
		const int e_idx = i * inst->dim.matrix_width + tx;
		const int p_idx = mem->p_zero + e_idx;
		const int n_idx = mem->lbn_zero + e_idx;

		double xi = P_ROW(ty)[p_idx];

		if(hiprand_uniform(&rnd_state) <= 0.5)
			xi = LB_ROW(ty)[p_idx] + curand_cauchy(&rnd_state) *
			     abs((LB_ROW(ty)[p_idx] - LBN_ROW(ty)[n_idx]));
		else
			xi = LBN_ROW(ty)[p_idx] + hiprand_normal(&rnd_state) *
			     abs((LB_ROW(ty)[p_idx] - LBN_ROW(ty)[n_idx]));

		xi = __dmul_rn(__double2uint_rn(DIV(xi, delta)), delta);
		xi = min(inst->parent_max, max(0., xi));
		P_ROW(ty)[p_idx] = xi;
	}
	__syncthreads();

	if(tx == 0 && ty == 0) {
		pso_ensure_constraints(inst, mem, &rnd_state);
	}

	inst->rnd_states[id] = rnd_state;
}
