#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "pso_rating.h"
#include "pso_memory.h"

__shared__ int MHEIGHT;
__shared__ int MWIDTH;

__shared__ double res[MATRIX_HEIGHT][MATRIX_WIDTH];
__shared__ double shrd_rating;
__shared__ double matrix_form;

__device__ inline void eval_set_res_matrix_to_zero()
{
	res[threadIdx.y][threadIdx.x] = 0.;
}

__device__ inline void eval_set_res_matrix_to_identity()
{
	if(threadIdx.x != threadIdx.y) {
		res[threadIdx.y][threadIdx.x] = 0.;
	} else {
		res[threadIdx.y][threadIdx.x] = 1.;
	}
}

__device__ inline void eval_copy_matrix_to_res(const struct pso_instance * const inst,
					       struct memory * const mem,
		    	    	    	       const int cmatrix)
{
	const int cstart = mem->p_zero + cmatrix * MWIDTH;
	res[ty][tx] = P_ROW(ty)[cstart + tx];
}

__device__ void eval_mul_inplace(const struct pso_instance * const inst,
				 struct memory         * const mem,
				 const int cmatrix)
{
	const int rows = MHEIGHT;
	const int cstart = mem->p_zero  + cmatrix * MWIDTH;

	double y, t;
	double c = 0;
	double sum = 0;

	/* result rows */
	#pragma unroll
	for(int i = 0; i < rows; i++) {
		y = __dmul_rn(res[ty][i], P_ROW(i)[cstart + tx]) - c;
		t = __dadd_rn(sum, y);
		c = (t - sum) - y;
		sum = t;
	}

	__syncthreads();
	res[ty][tx] = sum;
	__syncthreads();
}

__device__ const int* eval_interpret_rule(const struct pso_instance * const inst,
				    	  struct memory		    * const mem,
				    	  const int                 * rule)
{
	if(*rule == MUL_SEP)
		return rule;

	/*
	 * all multiplications are inplace,
	 * so we copy the first matrix to our result
	 */
	eval_copy_matrix_to_res(inst, mem, *rule);
	rule++;

	__syncthreads();

	for(; *rule != MUL_SEP; rule++) {
		eval_mul_inplace(inst, mem, *rule);
	}

	return rule;
}

__device__ void pso_result_rating(const struct pso_instance * const inst,
				  struct memory         * const mem)
{
	const int rows = MHEIGHT - 1;
	const int cols = MWIDTH  - 1;
	double rating = 0.;

	const double penalty = 1e9;

        if(ty == 0 && tx == 0) {
                switch(inst->cond_right) {
                case COND_UPPER_LEFT:
                        if((R_ROW(0)[mem->r_zero] - res[0][0]) < 1.f)
                                rating += penalty;
                        break;
                case COND_UPPER_RIGHT:
                        if((R_ROW(0)[mem->r_zero + cols] - res[0][cols]) < 1.f)
                                rating += penalty;
                        break;
                case COND_UPPER_LEFT_LOWER_RIGHT:
                        if((R_ROW(0)[mem->r_zero] - res[0][0]) < 1.f)
                                rating += penalty;

                        if((R_ROW(rows)[mem->r_zero + cols] - res[rows][cols]) < 1.f)
                                rating += penalty;
                        break;
                default:
                        rating += 2*penalty;
                        break;
                }

                if(inst->match == MATCH_ANY) {
                        if(rating == 0.)
                                matrix_form = 0.;

                        rating = 0.;
                }
        }
	__syncthreads();
	// keep only negative numbers
	res[ty][tx] = fabs(min(R_ROW(ty)[mem->r_zero + tx] - res[ty][tx], 0.));
	res[ty][tx] = __dmul_rn(res[ty][tx], res[ty][tx]);
	__syncthreads();

	double c = 0.0;
	double y, t;
	double sum;

	//only lines are processed
	if(tx == 0) {
		sum = res[ty][0];

		for(int i = 1; i < MWIDTH; i++) {
			y = res[ty][i] - c;
			t = sum + y;
			c = (t - sum) - y;
			sum = t;
		}

		res[ty][0] = sum;
	}
	__syncthreads();

	if(tx == 0 && ty == 0) {
		for(int i = 0; i < MHEIGHT; i++) {
			y = res[i][0] - c;
			t = rating + y;
			c = (t - rating) - y;
			rating = t;
		}

		shrd_rating += sqrtf(rating);
	}
	__syncthreads();
}

__global__ void pso_calc_res(struct pso_instance * const inst)
{
	__shared__ struct memory res_mem;

	const int* end = inst->rules + inst->rules_len - 1;
	const int* rules = inst->rules;
	struct memory * const mem = &res_mem;

	if(tx == 0 && ty == 0) {
		MHEIGHT = inst->dim.matrix_height;
		MWIDTH  = inst->dim.matrix_width;
		shrd_rating = 0.;
		matrix_form = 1e9;
		pso_init_mem(inst, &res_mem);
	}


	__syncthreads();
	uint8_t cur_rule = 0;

	do {
		eval_set_res_matrix_to_identity();

		rules++;
		rules = eval_interpret_rule(inst , mem, rules);

                __syncthreads();
                R_ROW(ty)[mem->r_zero + tx] = res[ty][tx];
		eval_set_res_matrix_to_identity();
                __syncthreads();

		rules++;
		rules = eval_interpret_rule(inst , mem, rules);
                __syncthreads();

		pso_result_rating(inst, mem);
		__syncthreads();

		cur_rule++;
		__syncthreads();
	} while(rules != end);

	__syncthreads();

	if(tx == 0 && ty == 0) {
		if(inst->match == MATCH_ANY)
			shrd_rating += matrix_form;

		res_mem.p_rat[blockIdx.y] = shrd_rating;
	}
}
