#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "instance.h"
#include "config.h"

#include "evo_memory.cu"

__device__ void double_memcpy(double* to, double* from, int size)
{
	while(size--) {
		*to = *from;
		to++;
		from++;
	}
}

__device__ void copy_child_to_parent(struct instance * const inst,
				     struct memory   * const mem,
				     const int child,
				     const int parent)
{
	const int cstart = child * inst->width_per_inst;
	const int pstart = parent * inst->width_per_inst;
	const int rows = MATRIX_HEIGHT;

	for(int r = 0; r < rows; r++) {
		double* const prow = P_ROW(r);
		double* const crow = C_ROW(r);

		double_memcpy(&(prow[pstart]),
		              &(crow[cstart]),
		              inst->width_per_inst);
	}
}

#include "evo_recombination.cu"
#include "evo_adaptive_gauss_mutation.cu"
#include "evo_selection.cu"

__global__ void evo_kernel(struct instance *inst, int flag)
{
	const int id = get_thread_id();

	/* copy global state to local mem for efficiency */
	hiprandState rnd_state = inst->rnd_states[id];

	struct memory mem;
	evo_init_mem(inst, &mem);

	int p_sel[2];
	double* sparam = get_sparam_arr(inst);

	const int tx = threadIdx.x;

	if(flag == 0) {
		evo_recomb_selection(inst, &rnd_state, p_sel);
		evo_recombination(inst, &mem, &rnd_state, p_sel);
		evo_mutation(inst, &mem, &rnd_state, &sparam[tx]);
	} else {
		evo_parent_selection_turnier(inst, &mem, &rnd_state, 3);
		__syncthreads();

		/* Parallel copy of memory */
		if(threadIdx.x < inst->dim.parents) {
			copy_child_to_parent(inst, &mem,
					     (int)mem.c_rat[2 * tx + 1], tx);
			mem.p_rat[threadIdx.x] = mem.c_rat[2 * tx];
		}
	}

	/* backup rnd state to global mem */
	inst->rnd_states[id] = rnd_state;
}
