#include "hip/hip_runtime.h"
#include <limits.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "instance.h"
#include "config.h"

__device__ void float_memcpy(float* to, float* from, int size)
{
//	memcpy(to, from, size * sizeof(float));

	if(((unsigned long long)to   & 0x4) ||
	   ((unsigned long long)from & 0x4)) {
		while(size--) {
			*to = *from;
			to++;
			from++;
		}
	} else {
		long long *t = (long long*)to;
		long long *f = (long long*)from;
		int s = size >> 1;

		while(s--) {
			*t = *f;
			t++;
			f++;
		}

		if(size & 1) {
			to   = (float*)t;
			from = (float*)f;
			*to  = *from;
		}
	}
}

/* calculate the thread id for the current block topology */
inline __device__ int get_thread_id() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void setup_rnd_kernel(hiprandState* rnd_states,
				 int seed)
{
	int id = get_thread_id();

	/* 
         * Each thread get the same seed, 
         * a different sequence number and no offset. 
         */
	hiprand_init(seed, id, 0, &rnd_states[id]);
}

/*
 * Initialize the parent memory with random values.
 */
__global__ void setup_parent_kernel(struct instance *inst)
{
	if(threadIdx.x >= inst->dim.matrix_height)
		return;

	int id = get_thread_id();
	hiprandState rnd_state = inst->rnd_states[id];

	char* devPtr = (char*)inst->dev_parent.ptr;
	size_t pitch = inst->dev_parent.pitch;
	size_t slicePitch = pitch * inst->dim.matrix_height;

	int z = blockIdx.x;
	int y = threadIdx.x;

	char* slice = devPtr + z * slicePitch;
	float* row = (float*) (slice + y * pitch);

	for(int x = 0; x < inst->dim.parents * inst->width_per_inst; x++) {
		if(hiprand_uniform(&rnd_state) < MATRIX_TAKEN_POS) {
			row[x] = hiprand(&rnd_state) % (int)PARENT_MAX;
		} else {
			row[x] = 0;
		}
	}

	inst->rnd_states[id] = rnd_state;

	if(threadIdx.x != 0)
		return;

	const int matrices = inst->num_matrices * inst->dim.parents;

	if(inst->cond_left == COND_UPPER_LEFT) {
		y = 0;
		row = (float*) (slice + y * pitch);

		for(int i = 0; i < matrices; i++) {
			row[i * inst->dim.matrix_width] = 1;
		}
	} else if(inst->cond_left == COND_UPPER_RIGHT) {
		y = 0;
		row = (float*) (slice + y * pitch);

		for(int i = 0; i < matrices; i++) {
			int idx = i * inst->dim.matrix_width + (inst->dim.matrix_width - 1);
			row[idx] = 1;
		}
	} else if(inst->cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
		y = 0;
		row = (float*) (slice + y * pitch);
		for(int i = 0; i < matrices; i++) {
			row[i * inst->dim.matrix_width] = 1;
		}

		y = (inst->dim.matrix_height - 1);
		row = (float*) (slice + y * pitch);
		for(int i = 0; i < matrices; i++) {
			int idx = i * inst->dim.matrix_width + (inst->dim.matrix_width - 1);
			row[idx] = 1;
		}
	}
}

#define C_ROW(y) ((float*) (mem->c_slice + y * mem->c_pitch))
#define P_ROW(y) ((float*) (mem->p_slice + y * mem->p_pitch))
#define R_ROW(y) ((float*) (mem->r_slice + y * mem->r_pitch))

struct memory {
	size_t p_pitch;
	char  *p_slice;

	size_t c_pitch;
	char  *c_slice;

	int c_zero;
	int c_end;

	size_t r_pitch;
	char  *r_slice;

	int r_zero1;
	int r_zero2;
	int r_end1;
	int r_end2;

	float* c_rat;
	float* p_rat;
};

__device__ void evo_init_mem(struct instance* inst, struct memory *mem)
{
	char* p_dev_ptr = (char*)inst->dev_parent.ptr;
	size_t p_pitch = inst->dev_parent.pitch;
	size_t p_slice_pitch = p_pitch * inst->dim.matrix_height;
	char* p_slice = p_dev_ptr + blockIdx.x /* z */ * p_slice_pitch;

	char* c_dev_ptr = (char*)inst->dev_child.ptr;
	size_t c_pitch = inst->dev_child.pitch;
	size_t c_slice_pitch = c_pitch * inst->dim.matrix_height;
	char* c_slice = c_dev_ptr + blockIdx.x /* z */ * c_slice_pitch;

	char* r_dev_ptr = (char*)inst->dev_res.ptr;
	size_t r_pitch = inst->dev_res.pitch;
	size_t r_slice_pitch = r_pitch * inst->dim.matrix_height;
	char* r_slice = r_dev_ptr + blockIdx.x /* z */ * r_slice_pitch;

	mem->p_pitch = p_pitch;
	mem->p_slice = p_slice;
	mem->c_pitch = c_pitch;
	mem->c_slice = c_slice;
	mem->r_pitch = r_pitch;
	mem->r_slice = r_slice;

	/*
	 * each thread represent one child which has a
	 * defined pos in the matrix
	 */
	mem->c_zero = inst->width_per_inst * threadIdx.x;
	mem->c_end  = inst->width_per_inst * (threadIdx.x + 1);

	mem->r_zero1 = threadIdx.x * 2 * inst->dim.matrix_width;
	mem->r_end1  = mem->r_zero1 + inst->dim.matrix_width;
	mem->r_zero2 = mem->r_zero1 + inst->dim.matrix_width;
	mem->r_end2  = mem->r_zero2 + inst->dim.matrix_width;

	char* t_dev_ptr = (char*)inst->dev_crat.ptr;
	size_t t_pitch = inst->dev_crat.pitch;
	size_t t_slice_pitch = t_pitch * 1;
	char* t_slice = t_dev_ptr + blockIdx.x /* z */ * t_slice_pitch;
	mem->c_rat = (float*) (t_slice + 0 * t_pitch);

	t_dev_ptr = (char*)inst->dev_prat.ptr;
	t_pitch = inst->dev_prat.pitch;
	t_slice_pitch = t_pitch * 1;
	t_slice = t_dev_ptr + blockIdx.x /* z */ * t_slice_pitch;
	mem->p_rat = (float*) (t_slice + 0 * t_pitch);
}

/*
 * Select two parents for recombination. Selection is currently complete uniform.
 */
inline __device__ void evo_recomb_selection(struct instance *inst, hiprandState *rnd_state, int *sel)
{
	sel[0] = hiprand(rnd_state) % inst->dim.parents;
	sel[1] = hiprand(rnd_state) % inst->dim.parents;
}

/* A uniform crossover recombination. */
__device__ void evo_recombination(struct instance *inst, 
				  struct memory   *mem,
				  hiprandState *rnd_state,
				  int *sel)
{
	int rows = inst->dim.matrix_height;
	int cols = inst->width_per_inst;

	int p1   = sel[0] * inst->width_per_inst;
	int p2   = sel[1] * inst->width_per_inst;
	int cIdx = mem->c_zero;

	for(int r = 0; r < rows; r++) {
		float* c_row = C_ROW(r);
		float* p_row = P_ROW(r);

		for(int c = 0; c < cols; c++) {
			if(hiprand_uniform(rnd_state) > RECOMB_RATE) {
				c_row[cIdx + c] = p_row[p1 + c];
			} else {
				c_row[cIdx + c] = p_row[p2 + c];
			}
		}
	}
}

__device__ void evo_ensure_constraints(struct instance *inst,
				       struct memory   *mem)
{
	float* row = C_ROW(0);
	int end   = mem->c_end;

	int factor = (int)(1.f / inst->delta);
	if((factor * inst->delta) < 1.f)
		factor++;

	float val = factor * inst->delta;

	for(int start = mem->c_zero; start < end; start += inst->dim.matrix_width) {
		if(inst->cond_right == COND_UPPER_LEFT && row[start] < 1.f) {
			row[start] = val;
		} else if(inst->cond_right == COND_UPPER_RIGHT &&
			  row[start + inst->dim.matrix_width - 1] < 1.f)
		{
			row[start + inst->dim.matrix_width - 1] = val;
		} else if(inst->cond_right == COND_UPPER_LEFT_LOWER_RIGHT) {
			row = C_ROW(0);
			if(row[start] < 1.f)
				row[start] = val;

			row = R_ROW(inst->dim.matrix_height - 1);
			if(row[start + inst->dim.matrix_width - 1] < 1.f)
				row[start + inst->dim.matrix_width - 1] = val;
		}
	}
}

__device__ void evo_mutation(struct instance *inst,
			     struct memory   *mem,
			     hiprandState *rnd_state,
                             float *s_param)
{
	*s_param = *s_param * exp(hiprand_normal(rnd_state) / 1000);
	int rows = inst->dim.matrix_height;
	float delta = inst->delta;
	float tmp;

	for(int r = 0; r < rows; r++) {
		float* row = C_ROW(r);

		for(int c = mem->c_zero; c < mem->c_end; c++) {

			if(hiprand_uniform(rnd_state) > MUT_RATE)
				continue;

			tmp = row[c];
			tmp = tmp + (float)(hiprand_normal(rnd_state) * (*s_param));
			/* we want x * delta, where x is an int */  	
			tmp = ((unsigned long)(tmp / delta)) * delta;
			tmp = max(tmp, 0.0);
			tmp = min(PARENT_MAX, tmp);

			row[c] = tmp;
		}
	}

	evo_ensure_constraints(inst, mem);
}

__device__ void evo_parent_selection(struct instance *inst, struct memory *mem)
{
	int elems = 2 * inst->dim.childs * inst->dim.parents;
	float* arr = mem->c_rat;

	float key, child;

	/* insertion sort */
	for(int i = 2; i < elems; i+=2) {
		key   = arr[i];
		child = arr[i+1];

		int j = i - 2;
		while(j >=0 && arr[j] > key) {
			arr[j + 2] = arr[j];
			arr[j + 3] = arr[j+1];
			j = j - 2;
		}
		arr[j + 2] = key;
		arr[j + 3] = child;
	}
}

__device__ volatile int cont = 1;


__device__ void copy_child_to_parent(struct instance *inst,
				     struct memory   *mem,
				     int child,
				     int parent)
{
	int cstart = child * inst->width_per_inst;
	int pstart = parent * inst->width_per_inst;
	int rows = inst->dim.matrix_height;

	for(int r = 0; r < rows; r++) {
		float* prow = P_ROW(r);
		float* crow = C_ROW(r);

		float_memcpy(&(prow[pstart]),
		             &(crow[cstart]),
		             inst->width_per_inst);
	}
}

/* extern device functions can't be inlined, so include them */
//#include "ensure.cu"
#include "evo_rating.cu"

__global__ void evo_kernel(struct instance *inst)
{
	int id = get_thread_id();

	/* copy global state to local mem for efficiency */
	hiprandState rnd_state = inst->rnd_states[id];

	struct memory mem;
	evo_init_mem(inst, &mem);

	int p_sel[2];
	float s_param = 5.f; /* TODO: For every matrix? */

	while(cont && inst->rounds < 10000) {
		evo_recomb_selection(inst, &rnd_state, p_sel);

		evo_recombination(inst, &mem, &rnd_state, p_sel);
		evo_mutation(inst, &mem, &rnd_state, &s_param);

		mem.c_rat[2 * threadIdx.x]     = evo_calc_res(inst, &mem);
		mem.c_rat[2 * threadIdx.x + 1] = threadIdx.x;
		if(mem.c_rat[2 * threadIdx.x] == 0.f) {
			inst->res_child_block = blockIdx.x;
			inst->res_child_idx   = threadIdx.x;
		}

		__syncthreads();

		/*
		 * All threads should rated their results here.
		 * It's time to get the new parents :D
		 */
		if(threadIdx.x == 0) {
			if(blockIdx.x == 0) {
				inst->rounds++;
			}

			evo_parent_selection(inst, &mem);
			if(mem.c_rat[0] == 0.f) {
				cont = 0;
				inst->res_block = blockIdx.x;
				inst->res_parent = threadIdx.x;
			}
		}

		__syncthreads();

		/* Parallel copy of memory */
		if(threadIdx.x < inst->dim.parents) {
			copy_child_to_parent(inst, &mem,
					     (int)mem.c_rat[2 * threadIdx.x + 1],
					     threadIdx.x);
			mem.p_rat[threadIdx.x] = mem.c_rat[2 * threadIdx.x];
		}

		__syncthreads();
	}

	/* backup rnd state to global mem */
	inst->rnd_states[id] = rnd_state;
}

__global__ void evo_kernel_test(struct instance *inst)
{
	int id = get_thread_id();

	/* copy global state to local mem for efficiency */
	hiprandState rnd_state = inst->rnd_states[id];

	struct memory mem;
	evo_init_mem(inst, &mem);

	int p_sel[2];
	float s_param = 5.f; /* TODO: For every matrix? */

	evo_recomb_selection(inst, &rnd_state, p_sel);
	evo_recombination(inst, &mem, &rnd_state, p_sel);
	evo_mutation(inst, &mem, &rnd_state, &s_param);
	mem.c_rat[2 * threadIdx.x]     = evo_calc_res(inst, &mem);
	mem.c_rat[2 * threadIdx.x + 1] = threadIdx.x;

	__syncthreads();
	/*
	 * All threads should rated their results here.
	 * It's time to get the new parents :D
	 */
	if(threadIdx.x == 0) {
		if(blockIdx.x == 0) {
			inst->rounds++;
		}

		evo_parent_selection(inst, &mem);
		if(mem.c_rat[0] == 0.f) {
			cont = 0;
			inst->res_block = blockIdx.x;
			inst->res_parent = threadIdx.x;
		}
	}

	__syncthreads();

	while(inst->rounds < 10000) {
		if(threadIdx.x == 0 && blockIdx.x == 0) {
			inst->rounds++;
		}

		/* Parallel copy of memory */
		if(threadIdx.x < inst->dim.parents) {
			copy_child_to_parent(inst, &mem,
					     (int)mem.c_rat[2 * threadIdx.x + 1],
					     threadIdx.x);
			mem.p_rat[threadIdx.x] = mem.c_rat[2 * threadIdx.x];
//			int res = ensure_correct_copy(inst, &mem,
//					(int) mem.c_rat[2 * threadIdx.x + 1],
//					threadIdx.x);
//			if(res) {
//				atomicAdd(&(inst->isnan), 100000);
//			}
		}

		__syncthreads();
	}

	/* backup rnd state to global mem */
	inst->rnd_states[id] = rnd_state;
}
