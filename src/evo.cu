#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "instance.h"
#include "config.h"

/* calculate the thread id for the current block topology */
inline __device__ int get_thread_id() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void setup_rnd_kernel(hiprandState* rnd_states,
				 int seed)
{
	int id = get_thread_id();

	/* 
         * Each thread get the same seed, 
         * a different sequence number and no offset. 
         */
	hiprand_init(seed, id, 0, &rnd_states[id]);
}

/*
 * Initialize the parent memory with random values.
 */
__global__ void
setup_parent_kernel(struct instance *inst)
{
	int id = get_thread_id();
	hiprandState rnd_state = inst->rnd_states[id];

	char* devPtr = (char*)inst->dev_parent.ptr;
	size_t pitch = inst->dev_parent.pitch;
	size_t slicePitch = pitch * inst->dim.matrix_height;

	int z = blockIdx.x;
//	int x = threadIdx.x;
	int y = threadIdx.x;

	char* slice = devPtr + z * slicePitch;
	float* row = (float*) (slice + y * pitch);

	for(int x = 0; x < inst->dim.threads * inst->dim.matrix_width; x++) {
		if(hiprand_uniform(&rnd_state) < MATRIX_TAKEN_POS) {
			row[x] = hiprand(&rnd_state);
		}
	}

	inst->rnd_states[id] = rnd_state;

	if(threadIdx.x != 0)
		return;

	if(inst->cond_left == COND_UPPER_LEFT) {
		y = 0;
		row = (float*) (slice + y * pitch);
		row[0] = 1;
	} else if(inst->cond_left == COND_UPPER_RIGHT) {
		y = 0;
		row = (float*) (slice + y * pitch);
		int x = (inst->dim.matrix_width - 1);
		row[x] = 1;
	} else if(inst->cond_left == COND_UPPER_LEFT_LOWER_RIGHT) {
		y = 0;
		row = (float*) (slice + y * pitch);
		row[0] = 1;

		y = (inst->dim.matrix_height - 1);
		int x = (inst->dim.matrix_width - 1);
		row = (float*) (slice + y * pitch);
		row[x] = 1;
	}
}


__global__ void evo_kernel(struct instance *inst)
{
	int id = get_thread_id();

	/* copy global state to local mem for efficiency */
	hiprandState rnd_state = inst->rnd_states[id];

		

	int x = hiprand(&rnd_state);

	/* backup rnd state to global mem */
	inst->rnd_states[id] = rnd_state;
}
